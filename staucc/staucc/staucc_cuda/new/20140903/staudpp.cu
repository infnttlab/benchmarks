#include "hip/hip_runtime.h"
#include "staudpp.h"
//nvcc -Wall -I/home/cuda-5.0/samples/common/inc -gencode arch=compute_20,code=sm_20 staudpp.cu -o staudpp
//nvcc  -I/home/cuda-5.0/samples/common/inc -gencode arch=compute_20,code=sm_20 staudpp.cu -o staudpp

// Scalar parameters

__constant__ float stepMax;
float stepMax_h;
__constant__ unsigned tgtKind;
unsigned tgtKind_h;

// Vector parameters

float *Mfeed_h, *Mfeed;
float *molSize_h, *molSize;
float *membSize_h, *membSize;

unsigned *leftSide_h, *leftSide;
unsigned *rightSide_h, *rightSide;
	
int *var_h, *var;
int *varSend_h, *varSend;	// varSend used only if tgtMatrix used (tgtKind=0)

int *tgtVector_h, *tgtVector;
int *tgtMatrix_h, *tgtMatrix;

float *Cvector_h, *Cvector;

// Output variables

FILE *log_fPtr;
FILE **buffer_fPtr;

__constant__ bool buffBool;
bool buffBool_h;
__constant__ unsigned buffEvery;
unsigned buffEvery_h;
__constant__ unsigned buffRows;
unsigned buffRows_h;
__constant__ unsigned buffCols;
unsigned buffCols_h;
__constant__ float eps;
float eps_h;

unsigned *buffRowCounterAllSims_h, *buffRowCounterAllSims;
float *buffMAllSims_h, *buffMAllSims;

// Simulation variables

unsigned long *seed_h, *seed;
hiprandState *rngStatesAllSims;

unsigned *step_h, *step;
float *t_h, *t;

float *MAllSims_h, *MAllSims;
float *MbkpAllSims;
unsigned *k_ruleAllSims;
unsigned *criticalAllSims;
unsigned *orderAllSims;
float *R_probAllSims;
float *R_probCAllSims;
float *M_sendAllSims; // ex SHARED memory
float *t1AllSims; // ex SHARED memory
float *tauAllSims; // ex SHARED memory
float *FSAllSims; // ex SHARED memory
float *FSbkpAllSims;
float *tauSSAAllSims;
unsigned *flagSSAAllSims;
unsigned *flagStepAllSims;

float *tauPreMinAllSims;
float *tau1AllSims;
float *tau2AllSims;
float *a0cAllSims;
float *a0AllSims;

unsigned *HORAllSims; // ex float HOR
float *auxVec1AllSims; // ex float mu, unsigned (anche float va bene) current
float *auxVec2AllSims; // ex float sigma





///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////// DEVICE code /////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////          

__global__ void initRNG(hiprandState *rngStatesAllSims, unsigned long *seed) {
    
	// membrane partition among threads
	int numMemsPerThread = numMems / blockDim.x;

	hiprandState *rngStates = rngStatesAllSims + blockIdx.x*numMems;

	for (int m=0; m<numMemsPerThread; m++)
		hiprand_init(seed[blockIdx.x], m*blockDim.x + threadIdx.x, 0, &rngStates[m*blockDim.x + threadIdx.x]);
}

__global__ void dynamicStep (hiprandState *rngStatesAllSims, 
  float *buffMAllSims, unsigned *buffRowCounterAllSims,
  unsigned *step, float *MAllSims, float *t, 
  unsigned *leftSide, unsigned *rightSide,
  int *var, int *varSend,
  int *tgtVector, int *tgtMatrix,
  float *Cvector,
  float *Mfeed, float *molSize, float *membSize,
  unsigned *k_ruleAllSims, unsigned *criticalAllSims, unsigned *orderAllSims, float *R_probAllSims, float *R_probCAllSims,
  float *M_sendAllSims, float *t1AllSims, float *tauAllSims, float *FSAllSims,
  float *FSbkpAllSims, 
  float *tauSSAAllSims, 
  float *MbkpAllSims,
  unsigned *flagSSAAllSims, unsigned *flagStepAllSims,
  float *tauPreMinAllSims, float *tau1AllSims, float *tau2AllSims, float *a0cAllSims, float *a0AllSims,
  unsigned *HORAllSims, float *auxVec1AllSims, float *auxVec2AllSims) {
  	  
  unsigned s = blockIdx.x;
  unsigned bDim = blockDim.x;
  unsigned thIdx = threadIdx.x;

  // membrane partition among threads
  unsigned numMemsPerThread = numMems / bDim;

  // per-thread scalar variables

  unsigned localStep;
  float localT;

  int m;
  int r, c, e;

  float aux1; // ex float a0cTmp, float numR, float maxTmp, unsigned (anche float va bene) currentTmp	       
  float aux2; // ex float minNumR, float tau1Tmp, unsigned HORTmp
  // anziche' avere unsigned HORTmp e float HOR ho fatto il contrario per avere meno variabili
  // tanto la conversione float -> unsigned dovrebbe farla lo stesso...

  //float rnd;
  float rnd_t, alpha;

  float splitShared; // ex t3, cn, globmintau, minFS

  hiprandState *rngStates = rngStatesAllSims + s*numMems;

  float *buffM = buffMAllSims + s*numMems*buffRows*buffCols;
  unsigned *buffRowCounter = buffRowCounterAllSims + s*numMems;

  unsigned *k_rule = k_ruleAllSims + s*numMems*numReacts;
  unsigned *critical = criticalAllSims + s*numMems*numReacts;
  unsigned *order = orderAllSims + s*numMems*numReacts;
  float *R_prob = R_probAllSims + s*numMems*numReacts;
  float *R_probC = R_probCAllSims + s*numMems*numReacts;

  float *M_send = M_sendAllSims + s*numMems*numSpecs;
  float *t1 = t1AllSims + s*numMems;
  float *tau = tauAllSims + s*numMems;
  float *FS = FSAllSims + s*numMems;

  float *FSbkp = FSbkpAllSims + s*numMems;
  float *tauSSA = tauSSAAllSims + s*numMems;
  unsigned *flagSSA = flagSSAAllSims + s*numMems;
  unsigned *flagStep = flagStepAllSims + s*numMems;
  float *tauPreMin = tauPreMinAllSims + s*numMems;
  float *tau1 = tau1AllSims + s*numMems;
  float *tau2 = tau2AllSims + s* numMems;
  float *a0c = a0cAllSims + s*numMems;
  float *a0 = a0AllSims + s*numMems;

  float *Mbkp = MbkpAllSims + s*numMems*numSpecs;
  float *M = MAllSims + s*numMems*numSpecs;
  unsigned *HOR = HORAllSims + s*numMems*numSpecs;   

  float *auxVec1 = auxVec1AllSims + s*blockDim.x*numSpecs;	   
  float *auxVec2 = auxVec2AllSims + s*blockDim.x*numSpecs;	

  ////////////////////////////////// Elaboration:

  localStep = 0;
  localT = 0.0f;

  // ex getFS
  for (m=0; m<numMemsPerThread; m++) {
  	  //if(m*bDim + thIdx > numMems) break;
  	  FS[m*bDim + thIdx] =  membSize[m*bDim + thIdx];
  	  for (c=0; c<numSpecs; c++) 
  		  FS[m*bDim + thIdx] -= (M[c*numMems+m*bDim+thIdx] * molSize[c*numMems+m*bDim+thIdx]);
  }

  while (localT<stepMax) {


  	  /////////////////////////////////////////////////////////
  	  // ex dpp_step1

  	  for (m=0; m<numMemsPerThread; m++) {
  		  FSbkp[m*bDim + thIdx] = FS[m*bDim + thIdx];
  		  for (c=0; c<numSpecs; c++) {
  			  Mbkp[c*numMems+m*bDim+thIdx] = M[c*numMems+m*bDim+thIdx];
  			  if (Mfeed[c*numMems+m*bDim+thIdx]>0)
  				  M[c*numMems+m*bDim+thIdx] = Mfeed[c*numMems+m*bDim+thIdx];
  		  }
  	  }

  	  ////////////////////////////////////////////////////////
  	  // ex rule_max_app()

  	  for (m=0; m<numMemsPerThread; m++) {
  		  aux1 = 0.0f;
  		  for (r=0; r<numReacts; r++) {
  			  aux2 = FLT_MAX;
  			  for (c=0; c<numSpecs; c++) {
  				  if (leftSide[r*numMems*numSpecs + c*numMems+m*bDim+thIdx] != 0)
  					  aux1 = M[c*numMems+m*bDim+thIdx] / (float)leftSide[r*numMems*numSpecs + c*numMems+m*bDim+thIdx];
  				  if (aux1<aux2)
  					  aux2 = aux1;
  			  }
  			  critical[r*numMems+m*bDim+thIdx] = (aux2>0.0f && aux2<10.0f);
  		  }
  	  }

  	  ////////////////////////////////////////////////////////
  	  // ex rule_prob()

  	  // ex getFS
  	  for (m=0; m<numMemsPerThread; m++) {
  		  FS[m*bDim + thIdx] = membSize[m*bDim + thIdx];
  		  for (c=0; c<numSpecs; c++) 
  			  FS[m*bDim + thIdx] -= (M[c*numMems+m*bDim+thIdx] * molSize[c*numMems+m*bDim+thIdx]);
  	  }

  	  for (m=0; m<numMemsPerThread; m++) {
  		  for (r=0; r<numReacts; r++) {
  			  R_prob[r*numMems+m*bDim+thIdx] = Cvector[r*numMems+m*bDim+thIdx];
  			  if (tgtVector[r*numMems+m*bDim+thIdx] == -1) {
  			     R_prob[r*numMems+m*bDim+thIdx] /= FS[m*bDim + thIdx];
  			     R_prob[r*numMems+m*bDim+thIdx] /= membSize[m*bDim + thIdx];
  			  }
  			  for (c=0; c<numSpecs; c++) {
  				  switch (leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]) {
  					  case 0:
  						  break;
  					  case 1:
  						  R_prob[r*numMems+m*bDim+thIdx] *= M[c*numMems+m*bDim+thIdx];
  						  break;
  					  case 2:
  						  R_prob[r*numMems+m*bDim+thIdx] *= (M[c*numMems+m*bDim+thIdx] * (M[c*numMems+m*bDim+thIdx] - 1)) * 0.5f;
  						  break;
  					  default:
  						  for (e=1; e<=leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]; e++)
  							  R_prob[r*numMems+m*bDim+thIdx] *= (M[c*numMems+m*bDim+thIdx] - e + 1.0f) / e;
  						  break;
  				  }
  			  }
  		  }
  	  }

  	  ////////////////////////////////////////////////////////

  	  for (m=0; m<numMemsPerThread; m++) {
  		  a0[m*bDim + thIdx] = 0.0f;
  		  for (r=0; r<numReacts; r++)
  			   a0[m*bDim + thIdx] += R_prob[r*numMems+m*bDim+thIdx];
  	  }

  	  for (m=0; m<numMemsPerThread; m++) {

  		  if (flagSSA[m*bDim + thIdx] == 0) {
  			  if (a0[m*bDim + thIdx]>0.0f) {

  				  ////////////////////////////////////////////////////////
  				  // ex get_HOR()

  				  for (r=0; r<numReacts; r++) {
  					  order[r*numMems+m*bDim+thIdx] = 0;
  					  for (c=0; c<numSpecs; c++)
  						  order[r*numMems+m*bDim+thIdx] += leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  				  }

  				  for (c=0; c<numSpecs; c++) {
  					  HOR[c*numMems+m*bDim+thIdx] = 0;
  					  auxVec1[c*bDim+thIdx] = 0.0f;
  					  for (r=0; r<numReacts; r++) {
  					  
  						  aux2 = 0.0f;
  						  aux1 = 0.0f;
  						  if (leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]!=0) {
  							  switch (order[r*numMems+m*bDim+thIdx]) {
  							     case 1:
  								     aux2  = 1.0f;
  								     aux1 = 1.0f;
  								     break;
  							     case 2:
  								     if (leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] == 1) {
  									     aux2  = 2.0f;
  									     aux1 = 2.0f;
  								     } else {
  									     aux2  = 2.0f + (1.0f / (M[c*numMems+m*bDim+thIdx] - 1.0f));
  									     aux1 = 3.0f;
  								     }
  								     break;
  							     case 3:
  								     if (leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] == 1) {
  									     aux2  = 3.0f;
  									     aux1 = 4.0f;
  								     } else if (leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] == 2) {
  									     aux2  = 1.5f * (2.0f + (1.0f / (M[c*numMems+m*bDim+thIdx] - 1.0f)));
  									     aux1 = 5.0f;
  								     } else {
  									     aux2  = 3.0f + (1.0f / (M[c*numMems+m*bDim+thIdx] - 1.0f)) + (2.0f / (M[c*numMems+m*bDim+thIdx] - 2.0f));
  									     aux1 = 6.0f;
  								     }
  								     break;
  							     default: // shouldn't be reached
  								     break;
  							  }
  						  }
  						  if (aux1 > auxVec1[c*bDim+thIdx]) {
  							  auxVec1[c*bDim+thIdx] = aux1;
  							  HOR[c*numMems+m*bDim+thIdx] = (unsigned) aux2;
  						  }
  					  }
  				  }

  			  ////////////////////////////////////////////////////////
  			  // ex get_tau1()

  			  for (c=0; c<numSpecs; c++) {
  				  auxVec1[c*bDim+thIdx] = 0.0f;
  				  auxVec2[c*bDim+thIdx] = 0.0f;
  				  for (r=0; r<numReacts; r++) 
  					  if (critical[r*numMems+m*bDim+thIdx] == 0) {
  						  auxVec1[c*bDim+thIdx] +=	(float)var[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]    * R_prob[r*numMems+m*bDim+thIdx];
  						  auxVec2[c*bDim+thIdx] += powf((float)var[r*numMems*numSpecs + c*numMems + m*bDim + thIdx],2) * R_prob[r*numMems+m*bDim+thIdx];
  					  }
  			  }

  			  aux2 = FLT_MAX;
  			  tau1[m*bDim + thIdx] = FLT_MAX;
  			  for (c=0; c<numSpecs; c++) {
  				  //aux1 = fmaxf((0.03f*M[c*numMems+m*bDim+thIdx]) / HOR[c*numMems+m*bDim+thIdx], 1);
				  aux1 = fmaxf((eps*M[c*numMems+m*bDim+thIdx]) / HOR[c*numMems+m*bDim+thIdx], 1);
  				  if (auxVec1[c*bDim+thIdx]!=0 && auxVec2[c*bDim+thIdx]!=0.0f)
  					  aux2 = fminf(aux1/fabs(auxVec1[c*bDim+thIdx]), powf(aux1,2)/auxVec2[c*bDim+thIdx]);
  				  tau1[m*bDim + thIdx] = fminf(aux2, tau1[m*bDim + thIdx]);
  			  }

  			  for (r=0; r<numReacts; r++) 
  				  k_rule[r*numMems+m*bDim+thIdx] = 0;

  				  for (r=0; r<numReacts; r++)
  					  if (critical[r*numMems+m*bDim+thIdx]==1)
  						  aux1 += R_prob[r*numMems+m*bDim+thIdx];
  				  do rnd_t = hiprand_uniform(&rngStates[m*bDim + thIdx]);
  				  while (rnd_t==1.0f);
  				  tau2[m*bDim + thIdx] = logf(1.0f/rnd_t)/aux1;

  				  if (tau1[m*bDim + thIdx]<tau2[m*bDim + thIdx]) {
  					  flagStep[m*bDim + thIdx] = 2;
  					  tau[m*bDim + thIdx] = tau1[m*bDim + thIdx];
  				  } else {
  					  flagStep[m*bDim + thIdx] = 3;
  					  tau[m*bDim + thIdx] = tau2[m*bDim + thIdx];
  					  a0c[m*bDim + thIdx] = aux1;
  					  tauPreMin[m*bDim + thIdx] = tau2[m*bDim + thIdx];
  				  }

  			  ////////////////////////////////////////////////////////

  		   } else {
  		      flagStep[m*bDim + thIdx] = 4;
  		      tau[m*bDim + thIdx] = FLT_MAX;
  		   }
  	    } else {
  		  flagStep[m*bDim + thIdx] = 5;
  		  tau[m*bDim + thIdx] = tauSSA[m*bDim + thIdx];
  	    }

  	  }

  	  ////////////////////////////////////////////////////////////////////////////////////

  	  __syncthreads();
  	  
  	  splitShared = FLT_MAX;	  
  	  for (m=0; m<numMems; m++)
  		  if (tau[m]<splitShared)
  			  splitShared = tau[m];

  	  for (m=0; m<numMemsPerThread; m++)
  		  tau[m*bDim + thIdx] = splitShared;

  	  ////////////////////////////////////////////////////////////////////////////////////
  	  // dpp_step2

  	  ////////////////////////////////////////////////////////
  	  // ex get_tau2()

  	  for (m=0; m<numMemsPerThread; m++) {
  		  
  		  for (c=0; c<numSpecs; c++)
  			  M_send[c*numMems+m*bDim+thIdx] = 0.0f;

  		  if (flagStep[m*bDim+thIdx]==2)
  			  for (r=0; r<numReacts; r++)
  				  if (critical[r*numMems+m*bDim+thIdx] == 0.0f)
  					  k_rule[r*numMems+m*bDim+thIdx] = hiprand_poisson(&rngStates[m*bDim+thIdx], tau[m*bDim+thIdx] * R_prob[r*numMems+m*bDim+thIdx]);
  		    
  		  if (flagStep[m*bDim+thIdx]==3) {
  			  for (r=0; r<numReacts; r++) {
  				  if (critical[r*numMems+m*bDim+thIdx] == 0.0f) {
  					  k_rule[r*numMems+m*bDim+thIdx] = hiprand_poisson(&rngStates[m*bDim+thIdx], tau[m*bDim+thIdx] * R_prob[r*numMems+m*bDim+thIdx]);
  					  R_probC[r*numMems+m*bDim+thIdx] = 0.0f;
  				  } else
  					  R_probC[r*numMems+m*bDim+thIdx] = R_prob[r*numMems+m*bDim+thIdx];
  			  }
  		  
  			  r=0;
  			  if (tauPreMin[m*bDim+thIdx]==tau[m*bDim+thIdx]) {
  				  do rnd_t = hiprand_uniform(&rngStates[m*bDim+thIdx]);
  				  while (rnd_t==1.0f);
  				  rnd_t *= a0c[m*bDim+thIdx];
  				  alpha = R_probC[r*numMems+m*bDim+thIdx];
  				  while (alpha<rnd_t) {
  					  r++;
  					  alpha += R_probC[r*numMems+m*bDim+thIdx];
  				  }
  				  k_rule[r*numMems+m*bDim+thIdx] = 1;
  			  }
  		  }

  		  ////////////////////////////////////////////////////////

  		  if (flagStep[m*bDim+thIdx]==2 || flagStep[m*bDim+thIdx]==3 || (tauSSA[m*bDim+thIdx] == tau[m*bDim+thIdx] && (flagStep[m*bDim+thIdx] == 1 || flagStep[m*bDim+thIdx] == 5)) ) {
  			  flagSSA[m*bDim+thIdx] = 0;
  			  if (tgtKind == 1) {
  				  for (r=0; r<numReacts; r++) {
  					  if (k_rule[r*numMems+m*bDim+thIdx]>0) {
  						  if (tgtVector[r*numMems+m*bDim+thIdx] == -1) // execute an internal rule
  							  for (c=0; c<numSpecs; c++)
  								  M[c*numMems+m*bDim+thIdx] += (float)var[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] * (float)k_rule[r*numMems+m*bDim+thIdx];
  						  else if (tgtVector[r*numMems+m*bDim+thIdx] == -2) { // execute a nondeterministic communication rule
  							  for (c=0; c<numSpecs; c++)
  								  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  							  for (e=0; e<k_rule[r*numMems+m*bDim+thIdx]; e++) {
  								  for (c=0; c<numSpecs; c++)
  									  atomicAdd(M_send + c*numMems + (int)ceilf((hiprand_uniform(&rngStates[m*bDim+thIdx])*numMems)), (float)rightSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]);
  							  }
  						  } else // execute a deterministic communication rule
  							  for (c=0; c<numSpecs; c++) {
  								  atomicAdd(M_send + c*numMems + tgtVector[r*numMems+m*bDim+thIdx], (float)k_rule[r*numMems+m*bDim+thIdx] * (float)rightSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]);
  								  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  							  }
  					  }
  				  }
  			  } else {
  				  for (r=0; r<numReacts; r++)
  					  if (k_rule[r*numMems+m*bDim+thIdx]>0)
  						  for (c=0; c<numSpecs; c++)
  							  atomicAdd(M_send + c*numMems + tgtMatrix[r*numMems*numSpecs + c*numMems + m*bDim + thIdx], (float)k_rule[r*numMems+m*bDim+thIdx] * (float)varSend[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] );
  				  // update the internal state of the process
  				  for (r=0; r<numReacts; r++)
  					  for (c=0; c<numSpecs; c++)
  						  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  			  }
  		  }

  		  if ( tauSSA[m*bDim+thIdx] != tau[m*bDim+thIdx] && (flagStep[m*bDim+thIdx] == 1 || flagStep[m*bDim+thIdx] == 5) )
  			  tauSSA[m*bDim+thIdx] -= tau[m*bDim+thIdx];

  	  }

  	  ////////////////////////////////////////////////////////////////////////////////////

  	  __syncthreads();

  	  // ex receive
  	  for (m=0; m<numMemsPerThread; m++)
  		  for (c=0; c<numSpecs; c++)
  			  M[c*numMems+m*bDim+thIdx] += M_send[c*numMems+m*bDim+thIdx];

  	  do {
  			  
  		  // ex getFS
  		  for (m=0; m<numMemsPerThread; m++) {
  			  FS[m*bDim + thIdx] = membSize[m*bDim + thIdx];
  			  for (c=0; c<numSpecs; c++) 
  				  FS[m*bDim + thIdx] -= (M[c*numMems+m*bDim+thIdx] * molSize[c*numMems+m*bDim+thIdx]);
  		  }

  		  __syncthreads();
  			  
  		  ////////////////////////////////////////////////////////
  		  // ex checkNegFS()

  		  splitShared = FLT_MAX;
  		  for (m=0; m<numMems; m++)
  			  if (FS[m]<splitShared)
  				  splitShared = FS[m];

  		  splitShared = (splitShared<0);
  	  
  		  ////////////////////////////////////////////////////////
  		  
  		  if (splitShared) {

  			  ////////////////////////////////////////////////////////////////////////////////////
  			  // dpp_step3

  			  for (m=0; m<numMemsPerThread; m++) {
  				  
  				  FS[m*bDim + thIdx] = FSbkp[m*bDim + thIdx];
  				  for (c=0; c<numSpecs; c++) {
  					  M[c*numMems+m*bDim+thIdx] = Mbkp[c*numMems+m*bDim+thIdx];
  					  M_send[c*numMems+m*bDim+thIdx] = 0.0f;
  				  }
  				  for (r=0; r<numReacts; r++) 
  					  k_rule[r*numMems+m*bDim+thIdx] = 0;

  				  tau[m*bDim + thIdx] /= 2.0f;
  				  tauSSA[m*bDim + thIdx] += tau[m*bDim + thIdx];

  				  if (flagStep[m*bDim + thIdx]==2 || flagStep[m*bDim + thIdx]==3)
  					  for(r=0; r<numReacts; r++)
  						  if (critical[r*numMems+m*bDim+thIdx] == 0.0f)
  							  k_rule[r*numMems+m*bDim+thIdx] = hiprand_poisson(&rngStates[m*bDim + thIdx], tau[m*bDim + thIdx] * R_prob[r*numMems+m*bDim+thIdx]);

  				  if (tgtKind == 1) {
  					  for (r=0; r<numReacts; r++) {
  						  if (k_rule[r*numMems+m*bDim+thIdx]>0) {
  							  if (tgtVector[r*numMems+m*bDim+thIdx] == -1) // execute an internal rule
  								  for (c=0; c<numSpecs; c++)
  									  M[c*numMems+m*bDim+thIdx] += (float)var[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] * (float)k_rule[r*numMems+m*bDim+thIdx];
  							  else if (tgtVector[r*numMems+m*bDim+thIdx] == -2) { // execute a nondeterministic communication rule
  								  for (c=0; c<numSpecs; c++)
  									  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  								  for (e=0; e<k_rule[r*numMems+m*bDim+thIdx]; e++) {
  									  for (c=0; c<numSpecs; c++)
  										  atomicAdd(M_send + c*numMems + (int)ceilf((hiprand_uniform(&rngStates[m*bDim+thIdx])*numMems)), (float)rightSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]);
  								  }
  							  } else // execute a deterministic communication rule
  								  for (c=0; c<numSpecs; c++) {
  									  atomicAdd(M_send + c*numMems + tgtVector[r*numMems+m*bDim+thIdx], (float)k_rule[r*numMems+m*bDim+thIdx] * (float)rightSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx]);
  									  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  								  }
  						  }
  					  }
  				  } else {
  					  for (r=0; r<numReacts; r++)
  						  if (k_rule[r*numMems+m*bDim+thIdx]>0)
  							  for (c=0; c<numSpecs; c++)
  								  atomicAdd(M_send + c*numMems + tgtMatrix[r*numMems*numSpecs + c*numMems + m*bDim + thIdx], (float)k_rule[r*numMems+m*bDim+thIdx] * (float)varSend[r*numMems*numSpecs + c*numMems + m*bDim + thIdx] );
  					  // update the internal state of the process
  					  for (r=0; r<numReacts; r++)
  						  for (c=0; c<numSpecs; c++)
  							  M[c*numMems+m*bDim+thIdx] -= (float)k_rule[r*numMems+m*bDim+thIdx] * (float)leftSide[r*numMems*numSpecs + c*numMems + m*bDim + thIdx];
  				  }
  			  }

  			  ////////////////////////////////////////////////////////////////////////////////////

  			  __syncthreads();

  			  
  			  // ex receive
  			  for (m=0; m<numMemsPerThread; m++)
  				  for (c=0; c<numSpecs; c++)
  					  M[c*numMems+m*bDim+thIdx] += M_send[c*numMems+m*bDim+thIdx];
  		  }

  	  } while (splitShared);

  	  localStep++;

  	  ////////////////////////////////////////////////////////////////////////////////////
  	  // dpp_step4

  	  for (m=0; m<numMemsPerThread; m++) {
  		  
  		  if (tau[m*bDim + thIdx] == FLT_MAX)
  			  tau[m*bDim + thIdx] = stepMax - localT + 1.0f;
  		  t1[m*bDim + thIdx] = localT + tau[m*bDim + thIdx];

  		  ////////////////////////////////////////////////////////////////////////////////////
  		  if (buffBool) {
  			  if (remainderf(localStep,buffEvery)==0.0f && buffRowCounter[m*bDim+thIdx]<buffRows) {
  				  
  				  buffM[buffRowCounter[m*bDim+thIdx]*numMems*buffCols + 0*numMems + m*bDim+thIdx] = t1[m*bDim + thIdx];
  				  for (c=0; c<numSpecs; c++)
  					  memcpy(buffM + buffRowCounter[m*bDim+thIdx]*numMems*buffCols + (c+1)*numMems + m*bDim, M + c*numMems + m*bDim, bDim * sizeof(float));
  				  buffRowCounter[m*bDim+thIdx]++;

  			  } else
  				  printf("buffer full!\n");
  		  }

  	  }

  	  __syncthreads();

  	  splitShared = 0;
  	  for (m=0; m<numMems; m++)
  			  if (t1[m]>splitShared)
  				  splitShared = t1[m];
      localT = splitShared;

  }

  step[s] = localStep;
  t[s] = localT;

}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// HOST code /////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////

void CUDAcall(hipError_t cudaResult, std::string variable) {

if (cudaResult != hipSuccess) {
	variable += ": ";
	variable += hipGetErrorString(cudaResult);
	throw std::runtime_error(variable);
}
}

/* Count the number of rows and columns of a binary file */
void uliRows_uliColumns_old(FILE *fPtr, unsigned *uliRows_h_Ptr, unsigned *uliCols_h_Ptr){
  long e;
  unsigned c=0, r=0;

  while ((e=getc(fPtr))!=EOF) {
  	if (e == '\t') c++;
  	if (e == '\n') break;
  }
  while ((e=getc(fPtr))!=EOF)
	if (e == '\n') r++;

  rewind(fPtr);
  *uliRows_h_Ptr = r + 1;
  *uliCols_h_Ptr = c; // + 1; vedi sotto
}

/* Count the number of rows and columns of a binary file */
void uliRows_uliColumns(FILE *fPtr, unsigned *uliRows_h_Ptr, unsigned *uliCols_h_Ptr){
  long e, pos;
  unsigned c=0, r=0;

  while ((e=getc(fPtr)) != '\n') ; //printf("%c",(char)c) ; //read #num\n

  pos = ftell(fPtr);
    
  while ((e=getc(fPtr))!=EOF) {
	if (e == '\t') c++;
  	if (e == '\n') break;
  }

  while (( (e=getc(fPtr))!=EOF) && (e != '#') )
  	if (e == '\n') r++;

  fseek(fPtr, pos, SEEK_SET);
  *uliRows_h_Ptr = r + 1;
  *uliCols_h_Ptr = c;// + 1; occhio che dopo l'ultimo elemento ci sia \t, altrimenti va messo +1 + 1;
}



/* Allocate and read an unsigned table from a file as a 1D vector */
unsigned* uLoad_table(FILE *fPtr, unsigned *uliRows_Ptr, unsigned *uliCols_Ptr) {
  unsigned j=0, i=0;
  unsigned *m;

  uliRows_uliColumns(fPtr, uliRows_Ptr, uliCols_Ptr);

  m = (unsigned *)malloc((*uliRows_Ptr)*(*uliCols_Ptr)*sizeof(unsigned) );
  for(j=0; j<*uliRows_Ptr; j++){
  	  for(i=0; i<*uliCols_Ptr-1; i++)
	  	fscanf(fPtr,"%i\t", &m[j*(*uliCols_Ptr)+i]);
  	  i = *uliCols_Ptr-1;
  	  fscanf(fPtr,"%i\n", &m[j*(*uliCols_Ptr)+i]);
  }
  return m;
}


/* Allocate and read an int table from a file as a 1D vector 
int* iLoad_table(FILE *fPtr, int mId) {
  int *m;
  unsigned  j=0, i=0;

  m = (int *) malloc( numReacts*numSpecs*sizeof(int) );
  for (j=0; j<numReacts; j++) {
	  for (i=0; i<numSpecs-1; i++)
		  fscanf(fPtr,"%d\t", &m[j*numSpecs+i]);
	  i=numSpecs-1;
	  fscanf(fPtr,"%d\n", &m[j*numSpecs+i]);
  }
  return m;
}
*/

/* Write the output matrix in the binary MULTI output file */
void write_matrix_bin(int m, int s) {
  
  if (buffBool_h) {
	  printf("sim %d mem %d: scrivo %ld righe (%ld bytes)\n", s, m, buffRowCounterAllSims_h[s*numMems + m], buffCols_h*buffRowCounterAllSims_h[s*numMems + m]*sizeof(float));
	  fwrite(&m, sizeof(unsigned),1,buffer_fPtr[s]);
	  for (unsigned c=0; c<buffCols_h; c++)
		fwrite(buffMAllSims_h + s*buffRows_h*buffCols_h*numMems + buffRowCounterAllSims_h[s*numMems + m]*buffCols_h*numMems + c*numMems + m, sizeof(float), 1, buffer_fPtr[s]);
	  buffRowCounterAllSims_h[s*numMems + m] = 0;
  } else {
	  fwrite(&m, sizeof(unsigned),1,buffer_fPtr[s]);
	  for (unsigned c=0; c<numSpecs; c++)
		  fwrite(MAllSims_h + s*numSpecs*numMems + c*numMems + m, sizeof(float), 1, buffer_fPtr[s]);
  }

}
/* Write the output matrix in the txt MULTI output file */
void write_matrix_txt(int m, int s) {
  
	if (buffBool_h) {
	  printf("sim %d mem %d: writing %d lines\n", s, m, buffRowCounterAllSims_h[s*numMems + m]);
	  for (unsigned r=0; r<buffRowCounterAllSims_h[s*numMems + m]; r++) {
		  fprintf(buffer_fPtr[s],"mem%4d: %4G\t", m, buffMAllSims_h[s*buffRows_h*buffCols_h*numMems + buffRowCounterAllSims_h[s*numMems + m]*buffCols_h*numMems + 0*numMems + m]);
		  for (unsigned c=0; c<numSpecs; c++)
			  fprintf(buffer_fPtr[s],"%4G\t", buffMAllSims_h[s*buffRows_h*buffCols_h*numMems + buffRowCounterAllSims_h[s*numMems + m]*buffCols_h*numMems + (c+1)*numMems + m]);
		  fprintf(buffer_fPtr[s],"\n");
	  }
	  buffRowCounterAllSims_h[s*numMems + m] = 0;
  } else {
	  fprintf(buffer_fPtr[s],"mem%4d: %4G\t", m, MAllSims_h[s*numSpecs*numMems + 0*numMems + m]);
	  for (unsigned c=1; c<numSpecs; c++)
		  fprintf(buffer_fPtr[s],"%4G\t", MAllSims_h[s*numSpecs*numMems + c*numMems + m]);
  }

  fprintf(buffer_fPtr[s],"\n");
}

// Read the first part of input files: LEFT/RIGHT_SIDE, TGT_VECTOR/MATRIX, INDEXES
void load_files1(int mId, unsigned** LS, unsigned** RS, int** TM, int** TV, FILE *flmatrix, FILE *frmatrix, FILE *ftgtmatrix, FILE *findexes) {
//  char input1[100], input2[100], app[20], file[60];
  unsigned int i, j;

  //sprintf(app,"%d", mId);

  unsigned rowsL, rowsR, colsL, colsR;

  fprintf(log_fPtr, "\n---> MEMBRANE: %d\n\n", mId);

  fprintf(log_fPtr,"Loaded stepMax:\n%g\n",stepMax_h);

  // Read the LEFT_SIDE input file with the mId or the common input file
/*
  sprintf(input1, "input/");
  sprintf(input2, "input/");
  strcpy(file,"left_side_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"left_side");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
	if ((fPtr = fopen (input2, "r")) == NULL)
		fprintf (log_fPtr, "\nCannot open left_side!\n");
*/

  LS[mId] = uLoad_table(flmatrix, &rowsL, &colsL);
//  fclose (fPtr);

  if (numSpecs != colsL)  fprintf(log_fPtr, "left cols %u != numSpecs %u --> using numSpecs\n", colsL, numSpecs);
  if (numReacts != rowsL) fprintf(log_fPtr, "left rows %u != numReacts %u --> using numReacts\n", rowsL, numReacts);

  // Print the LEFT HAND SIDE into the LOG file
  fprintf(log_fPtr,"\nLoaded left_side:\n");
  for (i=0; i<numReacts; i++) {
	for (j=0; j<numSpecs; j++)
		fprintf(log_fPtr,"%u\t",LS[mId][i*numSpecs+j]);
	fprintf(log_fPtr,"\n");
  }

  // Read the RIGHT_SIDE input file with the mId or the common input file
/*
  sprintf(input1, "input/");
  sprintf(input2, "input/");
  strcpy(file,"right_side_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"right_side");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
	if ((fPtr = fopen (input2, "r")) == NULL)
		fprintf (log_fPtr, "\nCannot open right_side!\n");
*/	
  RS[mId] = uLoad_table(frmatrix, &rowsR, &colsR);
//  fclose (fPtr);

  if (numSpecs != colsR)  fprintf(log_fPtr, "right cols %u != numSpecs %u --> using numSpecs\n", colsR, numSpecs);
  if (numReacts != rowsR) fprintf(log_fPtr, "right rows %u != numReacts %u --> using numReacts\n", rowsR, numReacts);

  // Print the RIGHT HAND SIDE into the LOG file
  fprintf(log_fPtr,"\nLoaded right_side:\n");
  for (i=0; i<numReacts; i++){
	  for (j=0; j<numSpecs; j++)
		  fprintf(log_fPtr,"%u\t",RS[mId][i*numSpecs+j]);
	  fprintf(log_fPtr,"\n");
  }

  // Read the TGT_VECTOR/TGT_MATRIX input file with the mId or the common input file
/*
  sprintf(input1, "input", numMems);
  sprintf(input2, "input", numMems);
  strcpy(file,"tgt_vector_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"tgt_vector");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
  	  if ((fPtr = fopen (input2, "r")) == NULL)
  		  fprintf (log_fPtr,"\nCannot open tgt_vector!\n");
*/


 	 
//  if (fPtr != NULL) {
  TV[mId] = (int *) malloc(numReacts * sizeof(int));
  tgtKind_h = 1;
  // Print the TGT_VECTOR into the LOG file

  fscanf(ftgtmatrix,"%*s\n"); //Read #num\n
  fprintf(log_fPtr,"\nLoaded tgt_vector:\n");

  for (j=0; j<numReacts-1; j++) {
  	fscanf(ftgtmatrix,"%d\t", &TV[mId][j]);
	fprintf(log_fPtr,"%d\t",TV[mId][j]);
  }

  fscanf(ftgtmatrix,"%d\n", &TV[mId][j]);
  fprintf(log_fPtr,"%d\n\n",TV[mId][j]);
  
/*
  	fclose (fPtr);
  } else {
	sprintf(input1, "input%d/", numMems);
    sprintf(input2, "input%d/", numMems);
    strcpy(file,"tgt_matrix_");
    strcat(file,app);
    strcat(input1,file);
    strcpy(file,"tgt_matrix");
    strcat(input2,file);
	if ((fPtr = fopen (input1, "r")) == NULL)
		if((fPtr = fopen (input2, "r")) == NULL)
  	        	fprintf (log_fPtr,"\nCannot open tgt_matrix\n");
  	        
  	if (fPtr != NULL) {
		TM[mId] = iLoad_table(fPtr,mId);
  	    fclose (fPtr);
  	    // Print the TGT_MATRIX in the LOG file
  	    fprintf(log_fPtr,"\nLoaded tgt_matrix:\n");
  	    for (i=0; i<numReacts; i++) {
  	    	for (j=0; j<numSpecs; j++)
  	    		fprintf(log_fPtr,"%d\t",TM[mId][i*numSpecs+j]);
	        fprintf(log_fPtr,"\n");
  	   }
	}
  	fprintf(log_fPtr,"\n");
  }
*/
  // Read the INDEXES input file with the mId or the common input file
  unsigned tmp;
  int indexes;
/*
  sprintf(input1, "input%d/", numMems);
  sprintf(input2, "input%d/", numMems);
  strcpy(file,"indexes_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"indexes");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
  	  if ((fPtr = fopen (input2, "r")) == NULL)
  		  fprintf (log_fPtr,"\nCannot open indexes!\n");
  		
  if(fPtr != NULL) {
*/

  // Print the indexes in the LOG file
  fprintf(log_fPtr,"\nLoaded indexes:\t");
  uliRows_uliColumns_old(findexes, &tmp, &buffCols_h);
  fprintf(log_fPtr,"\n-- %d --\n",buffCols_h);
  
  for (i=0; i<buffCols_h; i++) {
  	  fscanf(findexes, "%d\t", &indexes);
  	  fprintf(log_fPtr,"%d\t", indexes);
  }
  fscanf(findexes, "%d\n", &indexes);
  fprintf(log_fPtr,"%d\n\n", indexes);

  rewind(findexes);// THIS IS A SINGLE FILE, otherwise this part of the code has to be modified

  buffCols_h++; //perche'???

  fflush(log_fPtr);
//  }
}


// Read the second part of input files: C_VECTOR, M_0, M_FEED, SIZES
void load_files2(int mId, FILE * fm0, FILE *fcmatrix, FILE * fmfeed, FILE * fsize) {
//  FILE *fPtr;
  long double temp;
//  char input1[100], input2[100], app[20], file[60];
  unsigned int i,j;
/*
  sprintf(app,"%d", mId);

  // Read the C_VECTOR input file with the mId or the common input file
  sprintf(input1, "input%d/", numMems);
  sprintf(input2, "input%d/", numMems);
  strcpy(file,"c_vector_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"c_vector");
  strcat(input2,file);
*/
  fprintf(log_fPtr, "\n---> MEMBRANE: %d\n\n", mId);
/*
  if ((fPtr = fopen (input1, "r")) == NULL)
  	  if((fPtr = fopen (input2, "r")) == NULL)
  		  fprintf (log_fPtr,"\nCannot open c_vector!\n");
  	  
  if(fPtr != NULL) {
*/  

  fprintf(log_fPtr,"\nLoaded c_vector:\n");
  fscanf(fcmatrix,"%*s\n"); //Read #num\n
  
  for(j=0; j<numReacts-1; j++) {
	fscanf(fcmatrix, "%LG\t", &temp);
  	Cvector_h[j*numMems+mId] = (float) temp;
  	// Print the CONSTANTS VECTOR in the LOG file
  	fprintf(log_fPtr,"%g\t", Cvector_h[j*numMems+mId]);
  }

  fscanf(fcmatrix, "%LG\n", &temp);
  Cvector_h[j*numMems+mId] = (float) temp;
  // Print the CONSTANTS VECTOR in the LOG file
  fprintf(log_fPtr,"%g\n", Cvector_h[j*numMems+mId]);


//  }

  // Read the M_0 input file with the mId or the common input file

  unsigned tmp, colsM0;
/*
  sprintf(input1, "input%d/", numMems);
  sprintf(input2, "input%d/", numMems);
  strcpy(file,"M_0_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"M_0");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
  	  if((fPtr = fopen (input2, "r")) == NULL)
  		  fprintf (log_fPtr,"\nCannot open M_0!\n");

  if (fPtr != NULL) {
*/
  uliRows_uliColumns(fm0, &tmp, &colsM0);
  if(numSpecs != colsM0) printf("numSpecs %d != cols in M_0 %d --> using numSpecs\n", numSpecs, colsM0);
  fprintf(log_fPtr,"\nLoaded M_0:\n");
  for(i=0; i<numSpecs-1; i++) {
	fscanf(fm0, "%LG\t", &temp);
	MAllSims_h[0*numMems*numSpecs + i*numMems + mId] = (float) temp;
	for (unsigned s=0; s<numSims; s++) MAllSims_h[s*numMems*numSpecs + i*numMems + mId] = MAllSims_h[0*numMems*numSpecs + i*numMems + mId];
	// Print the MULTISET vector in the LOG file
	fprintf(log_fPtr,"%g\t", MAllSims_h[0*numMems*numSpecs + i*numMems + mId]);
  }
  fscanf(fm0, "%LG\n", &temp);
  MAllSims_h[0*numMems*numSpecs + i*numMems + mId] = (float) temp;
  for (unsigned s=0; s<numSims; s++) MAllSims_h[s*numMems*numSpecs + i*numMems + mId] = MAllSims_h[0*numMems*numSpecs + i*numMems + mId];
  fprintf(log_fPtr,"%g\n", MAllSims_h[0*numMems*numSpecs + i*numMems + mId]);
	  
	  
/*  fclose (fPtr);
  }
*/
  // Read the M_FEED input file with the mId or the common input file

/*
  sprintf(input1, "input%d/", numMems);
  sprintf(input2, "input%d/", numMems);
  strcpy(file,"M_feed_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"Mfeed");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
  if((fPtr = fopen (input2, "r")) == NULL)
  	  fprintf (log_fPtr,"\nCannot open Mfeed!\n");

  if(fPtr != NULL) {
*/

  fprintf(log_fPtr,"\nLoaded Mfeed:\n");
  fscanf(fmfeed,"%*s\n"); //Read #num\n
    
  for (i=0; i<numSpecs-1; i++) {
	  fscanf(fmfeed, "%LG\t", &temp);
	  Mfeed_h[i*numMems+mId] = (float) temp;
	  // Print the FEEDING MULTISET VECTOR in the LOG file
	  fprintf(log_fPtr,"%g\t", Mfeed_h[i*numMems+mId]);
  }
  fscanf(fmfeed, "%LG", &temp);
  Mfeed_h[i*numMems+mId] = (float) temp;
  // Print the FEEDING MULTISET VECTOR in the LOG file
  fprintf(log_fPtr,"%g\n\n", Mfeed_h[i*numMems+mId]);

/*	  
  	  fclose (fPtr);
  }
*/
  // Read the SIZES input file with the mId or the common input file
/*
  sprintf(input1, "input%d/", numMems);
  sprintf(input2, "input%d/", numMems);
  strcpy(file,"sizes_");
  strcat(file,app);
  strcat(input1,file);
  strcpy(file,"sizes");
  strcat(input2,file);
  if ((fPtr = fopen (input1, "r")) == NULL)
  	  if((fPtr = fopen (input2, "r")) == NULL)
  		  fprintf (log_fPtr,"\nCannot open sizes.txt\n");

  if(fPtr != NULL) {
*/
  fprintf(log_fPtr,"\nLoaded molSize:\n");
  fscanf(fsize,"%*s\n"); //Read #num\n  
  
  for(i=0; i<numSpecs-1; i++) {
	fscanf(fsize, "%LG\t", &temp);
	molSize_h[i*numMems+mId] = (float) temp;
	// Print the MOLECULAR SIZES in the LOG file
	fprintf(log_fPtr,"%g\t", molSize_h[i*numMems+mId]);
  }
  fscanf(fsize, "%LG\n", &temp);
  molSize_h[i*numMems+mId] = (float) temp;
  // Print the MOLECULAR SIZES in the LOG file
  fprintf(log_fPtr,"%g\n\n", molSize_h[i*numMems+mId]);
  
//  fprintf(log_fPtr,"\n");

  fprintf(log_fPtr,"\nLoaded membSize:\n");
  fscanf(fsize, "%LG\n", &temp);
  membSize_h[mId] = (float) temp;
  fprintf(log_fPtr,"%g\n",  membSize_h[mId]);
/*
  fclose (fPtr);
  }
*/
  fflush(log_fPtr);
}


// Set HOST data reading from input files
void CPUinitializeData(bool bBufferOut) {
  FILE *flmatrix, *frmatrix, *ftgtmatrix, *findexes;
  FILE *fm0, *fcmatrix, *fmfeed, *fsize;

  char file[100], input[100];

  printf("Number of simulations: %d\n", numSims);
  if (numSims < numSimsMin || numSims > numSimsMax)
     printf("\nSpecified number of simulations (%d) is invalid, must be between %d and %d.\n", numSims, numSimsMin, numSimsMax);

  buffBool_h = bBufferOut;

  printf("\nSeeds:\n");
  seed_h = (unsigned long *) malloc(numSims*sizeof(unsigned long));
  for (unsigned s=0; s<numSims; s++) {
  	  seed_h[s] = s + (unsigned long)time(NULL);
  	  printf("seed[%4d] = %lu\n", s, seed_h[s]);
      // Check requested seed is valid
      if (seed_h[s] == 0)
  	  printf("\nSpecified seed[%d] is invalid, must be non-zero.\n", s);
  }

  /* Read data common to all membranes */
  FILE *fPtr;
  long double temp;

  // Read time_max
  sprintf(input, "input/");
  strcpy(file,"time_max");
  strcat(input,file);
  if ((fPtr = fopen (input,"r")) == NULL)
      printf ("\nCannot open time_max\n");
  else {
      fscanf(fPtr,"%LG",&temp);
      stepMax_h = (float) temp;
      printf("\nTime max: %g\n", stepMax_h);
      fclose(fPtr);
  }

  // Read every
  sprintf(input, "input/");
  strcpy(file,"every");
  strcat(input,file);
  if ((fPtr = fopen (input, "r")) == NULL) {
  	  buffEvery_h = 1;
          buffRows_h = 10000;
  	  printf ("\nCannot open every - loading default %d and %d\n", buffEvery_h, buffRows_h);
  } else {
	fscanf(fPtr,"%d", &buffEvery_h);
      	fscanf(fPtr,"%d", &buffRows_h);
      	if (buffBool_h) {
  		printf ("\nOutput saved every: %d iteration(s)", buffEvery_h);
		printf("\nNumber of buffer lines: %d\n", buffRows_h);
      	} else  printf("\nOutput saved at final iteration\n");
	fclose (fPtr);
  }

  sprintf(input, "input/");
  strcpy(file,"eps");
  strcat(input,file);
  // Read eps
  if ((fPtr = fopen (input,"r")) == NULL)	  {
  	printf ("\nCannot open eps\n");
	eps_h = 0.03f;
  }
  else {
  	fscanf(fPtr,"%LG",&temp);
	eps_h = (float) temp;
  	printf("EPS: %g ",eps_h);
	fclose(fPtr);
  }


  // Read numMembranes
  unsigned numMemsFromFile;
  sprintf(input, "input/");
  strcpy(file,"numMembranes.txt");
  strcat(input,file);
  if ((fPtr = fopen (input,"r")) == NULL)
  	  printf ("\nCannot obtain the numbers of membranes, the file numMembranes.txt is required\n");
  else {
  	  fscanf(fPtr,"%d",&numMemsFromFile);
  	  fclose(fPtr);
  	  if (numMems != numMemsFromFile) printf("\nDefined number of membranes differs from that read from numMebranes\n");
  }

  /* HOST allocation (1st part) */

  // Output files

  if (buffBool_h) {
	buffRowCounterAllSims_h = (unsigned*) malloc(numSims * numMems * sizeof(unsigned));
	for (unsigned s=0; s<numSims; s++)
        	for (unsigned m=0; m<numMems; m++)
        		buffRowCounterAllSims_h[s*numMems + m] = 0;
  }

  buffer_fPtr	  = (FILE **)  malloc(numSims * sizeof(FILE*));

  // Temporary constants

  unsigned **unLeft_side2D_h, **unRight_side2D_h;
  int **iTgt_matrix2D_h, **iTgt_vector2D_h;
  unLeft_side2D_h  = (unsigned**) malloc(numMems * sizeof(void*));
  unRight_side2D_h = (unsigned**) malloc(numMems * sizeof(void*));
  iTgt_matrix2D_h  = (int **)	  malloc(numMems * sizeof(void*));
  iTgt_vector2D_h  = (int **)	  malloc(numMems * sizeof(void*));

  // Costants

  Mfeed_h	  = (float *) malloc(numMems*numSpecs * sizeof(float));
  molSize_h		  = (float *) malloc(numMems*numSpecs * sizeof(float));

  membSize_h			  = (float *) malloc(numMems * sizeof(float));

  // Variables

  t_h		  = (float *)	 malloc(numSims * sizeof(float));
  step_h	  = (unsigned *) malloc(numSims * sizeof(unsigned));

  MAllSims_h  = (float *)    malloc(numSims * numMems * numSpecs * sizeof(float));

  /* HOST assignments (1st part) */

  tgtKind_h = 0;

  char mem_sim[20];

  // Create a LOG file
  sprintf(file,"output%u_%u_%u/log.txt", numMems, threadsPerBlock, numSims);
  if ((log_fPtr = fopen (file,"w")) == NULL) {
  	  printf ("\nCannot write in the current directory - Log file\n");
  	  exit(0);
  }



// Read the first part of input files: LEFT/RIGHT_SIDE, TGT_VECTOR/MATRIX, INDEXES

  if((flmatrix = fopen ("input//L_matrix", "r")) == NULL){
	printf("Cannot open the left side\n");
	exit(0);
  }

  if((frmatrix = fopen ("input/R_matrix", "r")) == NULL){
	printf("Cannot open the right side\n");
	exit(0);
  }
  

  if((ftgtmatrix = fopen ("input/tgt_matrix", "r")) == NULL){
	printf("Cannot open the tgt matrix\n");
	exit(0);
  }


  if((findexes = fopen ("input/indexes", "r")) == NULL){
	printf("Cannot open indexes\n");
	exit(0);
  }

  /* For each membrane */
  for (unsigned m=0; m<numMems; m++) {
  	  // Read the first part of input files
  	  load_files1(m,unLeft_side2D_h, unRight_side2D_h, iTgt_matrix2D_h, iTgt_vector2D_h, flmatrix, frmatrix, ftgtmatrix, findexes);
  }


  fclose(flmatrix); fclose(frmatrix); fclose(ftgtmatrix); fclose(findexes);

  printf("Data 1 Acquired\n"); fflush(NULL);

  /* For each simulation */
  for (unsigned s=0; s<numSims; s++) {
  	  sprintf(mem_sim, "s%d.txt", s);
  		  
  	  // create a MULTI file labelled with the sId
  	  sprintf(file,"output%u_%u_%u/multi_",numMems, threadsPerBlock, numSims);
  	  strcat(file,mem_sim);
  	  if ((buffer_fPtr[s] = fopen (file,"w")) == NULL)
  		  printf ("\nCannot write in the current directory - Output file for sim = %u \n", s);
  		  
  	  // write on the MULTI file
  	  //fwrite(&buffCols,sizeof(unsigned),1,buffer_fPtr[s*numMems + m]);
  	  fprintf(buffer_fPtr[s], "%u\n", buffCols_h);
  }

  /* HOST allocation (2nd part) */

  /* Constants */
  leftSide_h	  = (unsigned*) malloc(numMems*numSpecs*numReacts * sizeof(unsigned));
  rightSide_h	  = (unsigned*) malloc(numMems*numSpecs*numReacts * sizeof(unsigned));
  var_h 	  = (int *)	malloc(numMems*numSpecs*numReacts * sizeof(int));
  varSend_h		  = (int *)	malloc(numMems*numSpecs*numReacts * sizeof(int));
  tgtMatrix_h	  = (int *)	malloc(numMems*numSpecs*numReacts * sizeof(int));
  tgtVector_h	  = (int *)	    malloc(numMems*numSpecs*numReacts * sizeof(int));
  Cvector_h		  = (float *)	malloc(numMems*numSpecs*numReacts * sizeof(float));

  if (buffBool_h)
  	  buffMAllSims_h   = (float *)  malloc(numSims * numMems*buffRows_h*buffCols_h * sizeof(float));

  for (unsigned i=0; i<numMems*numSpecs*numReacts; i++) {
  	  var_h[i] = 0;
  	  varSend_h[i] = 0;
  }

  /* For each membrane */

  if((fm0 = fopen ("input/M_0", "r")) == NULL){
	printf("Cannot open M_0\n");
	exit(0);
  }

  if((fcmatrix = fopen ("input/C_matrix", "r")) == NULL){
	printf("Cannot open the C values\n");
	exit(0);
  }


  if((fmfeed = fopen ("input/M_feed", "r")) == NULL){
	printf("Cannot open M_feed\n");
	exit(0);
  }


  if((fsize = fopen ("input/Size_matrix", "r")) == NULL){
	printf("Cannot open the size matrix\n");
	exit(0);
  }

  for (unsigned m=0; m<numMems; m++) {
  	  
  	  load_files2(m, fm0, fcmatrix, fmfeed, fsize);

  	  /* Convert iTgt_matrix2D_h and iTgt_vector2D_h into tgtMatrix_h and tgtVector_h */
  	  if (tgtKind_h == 1) {
  		  for (unsigned r=0; r<numReacts; r++)
  			  tgtVector_h[r*numMems+m] = iTgt_vector2D_h[m][r];
  		  free(iTgt_vector2D_h[m]);
  	  } else {
  		  for(unsigned r=0; r<numReacts; r++)
  			  for(unsigned c=0; c<numSpecs; c++)
  					  tgtMatrix_h[r*numMems*numSpecs + c*numMems + m] = iTgt_matrix2D_h[m][r*numSpecs + c];
  		  free(iTgt_matrix2D_h[m]);
  		  }

  	  /* Convert unLeft_side2D_h/unRight_side2D_h into leftSide_h/rightSide_h */
  	  /* Compute the VARIANTIONS MATRIX and print it into the LOG file */
  	  
  	  fprintf(log_fPtr, "\nComputed variations matrix var:\n");

  	  for(unsigned r=0; r<numReacts; r++) {
  		  for(unsigned c=0; c<numSpecs; c++) {
  			  leftSide_h [r*numMems*numSpecs + c*numMems + m] = unLeft_side2D_h[m][r*numSpecs+c];
  			  rightSide_h[r*numMems*numSpecs + c*numMems + m] = unRight_side2D_h[m][r*numSpecs+c];

  		      if (tgtKind_h == 1) {
  			  if(tgtVector_h[r*numMems+m] == -1) {
  				  var_h[r*numMems*numSpecs + c*numMems + m] -= leftSide_h [r*numMems*numSpecs + c*numMems + m];
  				  var_h[r*numMems*numSpecs + c*numMems + m] += rightSide_h[r*numMems*numSpecs + c*numMems + m];
  			  } else
  				  var_h[r*numMems*numSpecs + c*numMems + m] -= leftSide_h [r*numMems*numSpecs + c*numMems + m];
  		      } else {
  			  if (tgtMatrix_h[r*numMems*numSpecs + c*numMems + m] == m) {
  				  var_h[r*numMems*numSpecs + c*numMems + m] -= leftSide_h [r*numMems*numSpecs + c*numMems + m];
  				  var_h[r*numMems*numSpecs + c*numMems + m] += rightSide_h[r*numMems*numSpecs + c*numMems + m];
  				  varSend_h[r*numMems*numSpecs + c*numMems + m] = var_h[r*numMems*numSpecs + c*numMems + m];
  			  } else {
  				  var_h[r*numMems*numSpecs + c*numMems + m]	  -= leftSide_h [r*numMems*numSpecs + c*numMems + m];
  				  varSend_h[r*numMems*numSpecs + c*numMems + m] = rightSide_h[r*numMems*numSpecs + c*numMems + m];
  			  }
  		      }
  		      fprintf(log_fPtr,"%d\t",var_h[r*numMems*numSpecs + c*numMems + m]);
  		  }
  		  fprintf(log_fPtr,"\n");
  	  }
  	  free(unLeft_side2D_h[m]);
  	  free(unRight_side2D_h[m]);

  }
  free(unLeft_side2D_h); free(unRight_side2D_h); free(iTgt_matrix2D_h); free(iTgt_vector2D_h);
  fclose(fm0); fclose(fmfeed); fclose(fsize); fclose(fcmatrix);

  printf("Data 2 Acquired\n"); fflush(NULL);
}



float run(StopWatchInterface **timerOUT) {

  // Determine max threads per block
  int device = 0;
  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  cudaCheckErrors("get device properties");

  // Check requested size is valid
  if (threadsPerBlock < 1 || threadsPerBlock > static_cast<unsigned int>(deviceProperties.maxThreadsPerBlock))
     printf("specified block size (%d) is invalid, must be between %d and %d for device %d.\n", threadsPerBlock, 1, deviceProperties.maxThreadsPerBlock, device);

  hipFuncAttributes funcAttributes;
  dim3 block;
  dim3 grid;

  grid.x  = numSims;
  block.x = threadsPerBlock;

  // Get initRNG function properties and check the maximum block size
  hipFuncGetAttributes(&funcAttributes, reinterpret_cast<const void*>(initRNG));
  cudaCheckErrors("function get attributes");
  if (block.x > (unsigned)funcAttributes.maxThreadsPerBlock)
  	  printf("Block X dimension is too large for initRNG kernel");
  	  
  // Get dynamicStep function properties and check the maximum block size
  hipFuncGetAttributes(&funcAttributes, reinterpret_cast<const void*>(dynamicStep));
  cudaCheckErrors("function get attributes");
  if (block.x > (unsigned)funcAttributes.maxThreadsPerBlock)
  	  printf("Block X dimension is too large for dynamicStep kernel");

  // Check the maximum block size for device
  if (block.x > (unsigned)deviceProperties.maxThreadsDim[0])
  	  printf("Block X dimension for 'initRNG' kernel is too large for device");
  // Check the maximum grid size for device
  if (grid.x > (unsigned)deviceProperties.maxGridSize[0])
  	  printf("Grid X dimension is too large for device");

  // Allocate memory for RNG states 
  hiprandState *d_rngStates = 0;
  hipMalloc((void **)&d_rngStates, grid.x * numMems * sizeof(hiprandState));
  cudaCheckErrors("memory allocation on device for RNG states");
  
  // Initialize RNG
  initRNG<<<grid, block>>>(d_rngStates, seed);

  StopWatchInterface *timerIN = NULL;
  sdkCreateTimer(&timerIN);

  sdkStartTimer(timerOUT);
  sdkStartTimer(&timerIN);

 
  dynamicStep<<<grid, block>>>(d_rngStates,
  	  buffMAllSims, buffRowCounterAllSims,
  	  step, MAllSims, t,
  	  leftSide, rightSide,
  	  var, varSend,
  	  tgtVector, tgtMatrix,
  	  Cvector,
  	  Mfeed, molSize, membSize,
  	  k_ruleAllSims, criticalAllSims, orderAllSims, R_probAllSims, R_probCAllSims,
  	  M_sendAllSims, t1AllSims, tauAllSims, FSAllSims,
  	  FSbkpAllSims, 
  	  tauSSAAllSims, 
  	  MbkpAllSims,
  	  flagSSAAllSims, flagStepAllSims,
  	  tauPreMinAllSims, tau1AllSims, tau2AllSims, a0cAllSims, a0AllSims,
  	  HORAllSims, auxVec1AllSims, auxVec2AllSims);

  CUDAcall(hipDeviceSynchronize(),"dynamicStep");

  sdkStopTimer(timerOUT);
  sdkStopTimer(&timerIN);

  float elapsed = sdkGetAverageTimerValue(&timerIN);

  sdkDeleteTimer(&timerIN);

  return elapsed;
}

void GPUmallocData(){

	CUDAcall(hipMalloc((void **)&seed, numSims * sizeof(unsigned long)),"seed");

	if (buffBool_h) {
		CUDAcall(hipMalloc((void **)&buffMAllSims,  numSims * numMems*buffRows_h*buffCols_h * sizeof(float)),"buffM");
		CUDAcall(hipMalloc((void **)&buffRowCounterAllSims,  numSims * numMems * sizeof(unsigned)),"buffRowCounter");
	}
	
	// System costants

	CUDAcall(hipMalloc((void **)&Mfeed, numMems*numSpecs * sizeof(float)),"Mfeed");
	CUDAcall(hipMalloc((void **)&molSize, numMems*numSpecs * sizeof(float)),"molSize");
	CUDAcall(hipMalloc((void **)&membSize, numMems * sizeof(float)),"membSize");
	
	CUDAcall(hipMalloc((void **)&leftSide, numMems*numReacts*numSpecs * sizeof(unsigned)),"leftSide");
	CUDAcall(hipMalloc((void **)&rightSide, numMems*numReacts*numSpecs * sizeof(unsigned)),"rightSide");

	CUDAcall(hipMalloc((void **)&var, numMems*numReacts*numSpecs * sizeof(int)),"var");
	CUDAcall(hipMalloc((void **)&varSend, numMems*numReacts*numSpecs * sizeof(int)),"varSend");

	CUDAcall(hipMalloc((void **)&tgtVector, numMems*numReacts * sizeof(int)),"tgtVector");
	CUDAcall(hipMalloc((void **)&tgtMatrix, numMems*numReacts*numSpecs * sizeof(int)),"tgtMatrix");

	CUDAcall(hipMalloc((void **)&Cvector, numMems*numReacts * sizeof(float)),"Cvector");

	// System variables

	CUDAcall(hipMalloc((void**)&t,		numSims * sizeof(float)), "t");
	CUDAcall(hipMalloc((void**)&step,	numSims * sizeof(unsigned)), "step");
	CUDAcall(hipMalloc((void **)&MAllSims,	numSims * numMems * numSpecs * sizeof(float)),"MAllSims");

	CUDAcall(hipMalloc((void **)&k_ruleAllSims,   numSims * numMems*numReacts * sizeof(unsigned)),"uliK_rule");
	CUDAcall(hipMalloc((void **)&criticalAllSims, numSims * numMems*numReacts * sizeof(unsigned)),"uliCritical");
	CUDAcall(hipMalloc((void **)&orderAllSims,    numSims * numMems*numReacts * sizeof(unsigned)),"order");
	CUDAcall(hipMalloc((void **)&R_probAllSims,   numSims * numMems*numReacts * sizeof(float)),"R_prob");
	CUDAcall(hipMalloc((void **)&R_probCAllSims,  numSims * numMems*numReacts * sizeof(float)),"R_probC");
	
	// ex SHARED memory variables
	CUDAcall(hipMalloc((void**)&M_sendAllSims, numSims * numMems * numSpecs * sizeof(float)),"M_send");
	CUDAcall(hipMalloc((void**)&t1AllSims, numSims * numMems * sizeof(float)),"t1");
	CUDAcall(hipMalloc((void**)&tauAllSims, numSims * numMems * sizeof(float)),"tau");
	CUDAcall(hipMalloc((void**)&FSAllSims, numSims * numMems * sizeof(float)),"FS");
	//

	CUDAcall(hipMalloc((void**)&FSbkpAllSims, numSims * numMems * sizeof(float)),"FSbkp");
	CUDAcall(hipMalloc((void**)&tauSSAAllSims, numSims * numMems * sizeof(float)),"tauSSA");
	
	CUDAcall(hipMalloc((void**)&MbkpAllSims, numSims * numSpecs * numMems * sizeof(float)),"Mbkp");

	CUDAcall(hipMalloc((void**)&flagSSAAllSims, numSims * numMems * sizeof(unsigned)),"flagSSA");
	CUDAcall(hipMalloc((void**)&flagStepAllSims, numSims * numMems * sizeof(unsigned)),"flagStep");

	CUDAcall(hipMalloc((void**)&tauPreMinAllSims, numSims * numMems * sizeof(float)),"tauPreMin");
	CUDAcall(hipMalloc((void**)&tau1AllSims, numSims * numMems * sizeof(float)),"tau1");
	CUDAcall(hipMalloc((void**)&tau2AllSims, numSims * numMems * sizeof(float)),"tau2");
	CUDAcall(hipMalloc((void**)&a0cAllSims, numSims * numMems * sizeof(float)),"a0c");
	CUDAcall(hipMalloc((void**)&a0AllSims, numSims * numMems * sizeof(float)),"a0");

	CUDAcall(hipMalloc((void**)&HORAllSims, numSims * numSpecs * numMems * sizeof(unsigned)),"HOR");       
	CUDAcall(hipMalloc((void**)&auxVec1AllSims, numSims * threadsPerBlock * numSpecs * sizeof(float)),"auxVec1");         
	CUDAcall(hipMalloc((void**)&auxVec2AllSims, numSims * threadsPerBlock * numSpecs * sizeof(float)),"auxVec2");     

}
void GPUfreeData() {

	CUDAcall(hipFree(seed),"seed");
	
	if (buffBool_h) {
		CUDAcall(hipFree(buffMAllSims),"buffM");
		CUDAcall(hipFree(buffRowCounterAllSims),"buffRowCounter");
	}

	// System costants

	CUDAcall(hipFree(Mfeed),"Mfeed");
	CUDAcall(hipFree(molSize),"molSize");
	CUDAcall(hipFree(membSize),"membSize");
			
	CUDAcall(hipFree(leftSide),"leftSide");
	CUDAcall(hipFree(rightSide),"rightSide");
			
	CUDAcall(hipFree(var),"var");
	CUDAcall(hipFree(varSend),"varSend");
		
	CUDAcall(hipFree(tgtVector),"tgtVector");
	CUDAcall(hipFree(tgtMatrix),"tgtMatrix");

	CUDAcall(hipFree(Cvector),"Cvector");

	// System variables

	CUDAcall(hipFree(t),"t");
	CUDAcall(hipFree(step),"step");
				
	CUDAcall(hipFree(MAllSims),"MAllSims");	

	CUDAcall(hipFree(k_ruleAllSims),"k_rule");
	CUDAcall(hipFree(criticalAllSims),"critical");
	CUDAcall(hipFree(orderAllSims),"order");
	CUDAcall(hipFree(R_probAllSims),"R_prob");
	CUDAcall(hipFree(R_probCAllSims),"R_probC");
	CUDAcall(hipFree(M_sendAllSims),"M_send"); // ex SHARED memory
	CUDAcall(hipFree(t1AllSims),"t1"); // ex SHARED memory
	CUDAcall(hipFree(tauAllSims),"tau"); // ex SHARED memory
	CUDAcall(hipFree(FSAllSims),"FS"); // ex SHARED memory

	CUDAcall(hipFree(FSbkpAllSims),"FSbkp");
	CUDAcall(hipFree(tauSSAAllSims),"tauSSA");
	
	CUDAcall(hipFree(MbkpAllSims),"Mbkp");

	CUDAcall(hipFree(flagSSAAllSims),"flagSSA");
	CUDAcall(hipFree(flagStepAllSims),"flagStep");

	CUDAcall(hipFree(tauPreMinAllSims),"tauPreMin");
	CUDAcall(hipFree(tau1AllSims),"tau1");
	CUDAcall(hipFree(tau2AllSims),"tau2");
	CUDAcall(hipFree(a0cAllSims),"a0c");
	CUDAcall(hipFree(a0AllSims),"a0");

	CUDAcall(hipFree(HORAllSims),"HOR");       
	CUDAcall(hipFree(auxVec1AllSims),"auxVec1");          
	CUDAcall(hipFree(auxVec2AllSims),"auxVec2");    

}

void CPUfreeData() {

  free(seed_h);

  if (buffBool_h) {
  	  free(buffMAllSims_h);
  	  free(buffRowCounterAllSims_h);
  }

  // System costants
  			  
  free(membSize_h);
  		  
  free(leftSide_h);
  free(rightSide_h);
  		  
  free(var_h);
  free(varSend_h);
  	  
  free(tgtVector_h);
  free(tgtMatrix_h);
  		  
  free(Mfeed_h);
  free(molSize_h);
  		  
  free(Cvector_h);
  		  
  // System variables

  free(t_h);
  free(step_h);

  free(MAllSims_h);	  

  fclose(log_fPtr);
  for (unsigned s=0; s<numSims; s++) fclose(buffer_fPtr[s]);
}


void uploadData() {

	CUDAcall(hipMemcpy(seed, seed_h, numSims * sizeof(unsigned long), hipMemcpyHostToDevice),"seed");

	if (buffBool_h) {
		CUDAcall(hipMemcpy(buffMAllSims, buffMAllSims_h, numSims * numMems*buffRows_h*buffCols_h* sizeof(float), hipMemcpyHostToDevice),"buffM");
		CUDAcall(hipMemcpy(buffRowCounterAllSims,	buffRowCounterAllSims_h, numSims * numMems * sizeof(unsigned), hipMemcpyHostToDevice),"buffRowCounter");
	}

	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(buffEvery), 	&buffEvery_h, sizeof(unsigned), 0, hipMemcpyHostToDevice), "buffEvery");
	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(buffRows), 	&buffRows_h,  sizeof(unsigned), 0, hipMemcpyHostToDevice), "buffRows");
	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(buffCols), 	&buffCols_h,  sizeof(unsigned), 0, hipMemcpyHostToDevice), "buffCols");
	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(buffBool), 	&buffBool_h,  sizeof(bool),     0, hipMemcpyHostToDevice), "buffBool");
	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(eps), 	&eps_h,       sizeof(float),    0, hipMemcpyHostToDevice), "eps");
	
	
	// System costants

	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(stepMax), 		&stepMax_h,     sizeof(float),    0, hipMemcpyHostToDevice), "stepMax");
	CUDAcall(hipMemcpyToSymbol(HIP_SYMBOL(tgtKind), 	&tgtKind_h, 	 sizeof(unsigned), 0, hipMemcpyHostToDevice), "tgtKind");

	CUDAcall(hipMemcpy(leftSide, leftSide_h, numMems*numReacts*numSpecs *sizeof(unsigned), hipMemcpyHostToDevice),"leftSide");
	CUDAcall(hipMemcpy(rightSide, rightSide_h, numMems*numReacts*numSpecs * sizeof(unsigned), hipMemcpyHostToDevice),"rightSide");

	CUDAcall(hipMemcpy(var, var_h, numMems*numReacts*numSpecs * sizeof(int), hipMemcpyHostToDevice),"var");
	CUDAcall(hipMemcpy(varSend, varSend_h, numMems*numReacts*numSpecs * sizeof(int), hipMemcpyHostToDevice),"varSend");

	CUDAcall(hipMemcpy(tgtMatrix, tgtMatrix_h, numMems*numReacts*numSpecs * sizeof(int), hipMemcpyHostToDevice),"tgtMatrix");
	CUDAcall(hipMemcpy(tgtVector, tgtVector_h, numMems*numReacts * sizeof(int), hipMemcpyHostToDevice),"tgtVector");

	CUDAcall(hipMemcpy(Cvector, Cvector_h, numMems*numReacts * sizeof(float), hipMemcpyHostToDevice),"Cvector");

	CUDAcall(hipMemcpy(membSize, membSize_h, numMems * sizeof(float), hipMemcpyHostToDevice),"membSize");
	CUDAcall(hipMemcpy(Mfeed, Mfeed_h, numMems*numSpecs * sizeof(float), hipMemcpyHostToDevice),"Mfeed");
	CUDAcall(hipMemcpy(molSize, molSize_h, numMems*numSpecs * sizeof(float), hipMemcpyHostToDevice),"molSize");

	// System variables

	CUDAcall(hipMemcpy(MAllSims, MAllSims_h,		numSims * numMems * numSpecs * sizeof(float), hipMemcpyHostToDevice),"MAllSims");
	
	CUDAcall(hipMemset(orderAllSims, 0, numSims * numMems*numReacts * sizeof(unsigned)),"orderAllSims");
	CUDAcall(hipMemset(tauSSAAllSims, 0, numSims * numMems * sizeof(unsigned)),"tauSSAAllSims");
	CUDAcall(hipMemset(flagSSAAllSims, 0, numSims * numMems * sizeof(unsigned)),"flagSSAAllSims");

	CUDAcall(hipMemset(tauAllSims, 0, numSims * numMems * sizeof(float)),"tauAllSims");
}
void downloadData() {

	if (buffBool_h) {
		CUDAcall(hipMemcpy(buffMAllSims_h, buffMAllSims, numSims * numMems*buffRows_h*buffCols_h* sizeof(float), hipMemcpyDeviceToHost),"buffM");
		CUDAcall(hipMemcpy(buffRowCounterAllSims_h, buffRowCounterAllSims,	numSims * numMems * sizeof(unsigned), hipMemcpyDeviceToHost),"buffRowCounter");
	}

	// System variables

	CUDAcall(hipMemcpy(t_h, t,	numSims * sizeof(float), hipMemcpyDeviceToHost), "t");
	CUDAcall(hipMemcpy(step_h, step, numSims * sizeof(unsigned), hipMemcpyDeviceToHost), "step");

	CUDAcall(hipMemcpy(MAllSims_h, MAllSims,	numSims * numMems*numSpecs*sizeof(float), hipMemcpyDeviceToHost),"MAllSims");
}
void printResult() {

	for (unsigned m=0; m<numMems; m++)
		for (unsigned s=0; s<numSims; s++) {
			//write_matrix_bin(m, s);
			write_matrix_txt(m, s);
		}

}

int main(int argc, char **argv) {

  hipDeviceProp_t deviceProperties;

  // Get number of available devices
  int deviceCount = 0;
  hipError_t cudaResult = hipGetDeviceCount(&deviceCount);
  cudaCheckErrors("get device count");

  // Get device properties
  int device = 0;
  hipGetDeviceProperties(&deviceProperties, device);
  cudaCheckErrors("get device properties");

  // Check precision
  if (deviceProperties.major < 1 || (deviceProperties.major == 1 && deviceProperties.minor < 3))
  	  printf("device does not have float precision support");

  // Attach to GPU
  hipSetDevice(device);
  cudaCheckErrors("set device");

  bool bBufferOut = 0;

  char folder[500];
  std::sprintf(folder, "mkdir output%u_%u_%u", numMems, threadsPerBlock, numSims);
  system(folder);

  char prompt[500];
  std::sprintf(prompt, "output%u_%u_%u/prompt_s%u_m%u_buff%d.txt", numMems, threadsPerBlock, numSims, numSims, numMems, bBufferOut);
  /*
  if (std::freopen(prompt, "w", stdout) == NULL) {
  	      printf ("cannot open %s", prompt);
  	      exit(0);
  }
  */
  StopWatchInterface *timer_ALLOC_UPLOAD = NULL, *timer_FREE_DOWNLOAD = NULL;
  StopWatchInterface *timer_CALC = NULL;
  sdkCreateTimer(&timer_ALLOC_UPLOAD);
  sdkCreateTimer(&timer_FREE_DOWNLOAD);
  sdkCreateTimer(&timer_CALC);

  float *elapsedTime_CALC = (float*) malloc(numTrials * sizeof (float));
  unsigned *numSteps = (unsigned*) malloc(numTrials * numSims * sizeof (unsigned));
  unsigned *numStepsAllSims = (unsigned*) malloc(numTrials * sizeof (unsigned));
  float timePerStepMedium_CALC = 0;

  size_t varSize=0, parSize=0;

  for (unsigned trial=0; trial<numTrials; trial++) {
	CPUinitializeData(bBufferOut);
  	
  	sdkStartTimer(&timer_ALLOC_UPLOAD);
  	GPUmallocData();
  	uploadData();
  	sdkStopTimer(&timer_ALLOC_UPLOAD);

	//Main Function
  	elapsedTime_CALC[trial] = run(&timer_CALC);

  	sdkStartTimer(&timer_FREE_DOWNLOAD);
  	downloadData();
  	GPUfreeData();
  	sdkStopTimer(&timer_FREE_DOWNLOAD);

  	numStepsAllSims[trial] = 0;
  	for (unsigned s=0; s<numSims; s++) {
  	        numSteps[trial*numSims + s] = step_h[s];
  	        numStepsAllSims[trial] += numSteps[trial*numSims + s];
  	}

  	timePerStepMedium_CALC += elapsedTime_CALC[trial]/numStepsAllSims[trial];

  	printResult();

  	if (trial==0) {
  	        varSize = numSims * (3 * numMems*numReacts * sizeof(unsigned) +
  	        			2 * numMems*numReacts * sizeof(float) +
  	        			4 * numMems * numSpecs * sizeof(float) +
  	        			10 * numMems * sizeof(float) +
  	        			2 * numSims * numMems * sizeof(unsigned) +
  	        			1 * numSims * numSpecs * numMems * sizeof(unsigned));

  	        parSize = numSims * numMems * sizeof(hiprandState) + // rngStates
  	        			numMems * sizeof(unsigned) + // numReacts
  	        			numSims * sizeof(unsigned) + // step
  	        			numSims * numMems * numSpecs * sizeof(float) + // M
  	        			numSims * sizeof(float) + // t
  	        			2 * numMems*numReacts*numSpecs * sizeof(unsigned) + // leftSide,rightSide
  	        			2 * numMems*numReacts*numSpecs * sizeof(int) + // var, varSend
  	        			numMems*numReacts * sizeof(int) + // tgtVector
  	        			numMems*numReacts*numSpecs * sizeof(int) + // tgtMatrix
  	        			numMems*numReacts * sizeof(float) + // Cvector
  	        			2 * numMems * numSpecs * sizeof(float) + // Mfeed, molSize
  	        			numMems * sizeof(float); // membSize

  	        if (bBufferOut)
  	        	parSize += numSims * numMems*buffRows_h*(numSpecs+1) * sizeof(float) + // buffM
  	        			numSims * numMems * sizeof(unsigned); // buffRowCounter
  	        /*
  	        k_rule, critical, order: numSims * numMems*numReacts * sizeof(unsigned)
  	        R_prob, R_probC: numSims * numMems*numReacts * sizeof(float)
  	        M_send, Mbkp, auxVec1, auxVec2: numSims * numMems * numSpecs * sizeof(float)
  	        t1, tau, FS, FSbkp, tauSSA, tauPreMin, tau1, tau2, a0c, a0: numSims * numMems * sizeof(float)
  	        flagSSA, flagStep: numSims*numMems*sizeof(unsigned)
  	        HOR: numSims*numSpecs*numMems*sizeof(unsigned)  
  	        rngStates: numSims * numMems * sizeof(hiprandState)
  	        buffM: numSims * numMems*buffRows_h*(numSpecs+1) * sizeof(float)
  	        buffRowCounter: numSims * numMems * sizeof(unsigned)
  	        numReacts: numMems * sizeof(unsigned)
  	        step: numSims * sizeof(unsigned)
  	        M: numSims * numMems * numSpecs * sizeof(float)
  	        t: numSims * sizeof(float)
  	        leftSide: numMems*numReacts*numSpecs * sizeof(unsigned)
  	        rightSide: numMems*numReacts*numSpecs * sizeof(unsigned)
  	        var: numMems*numReacts*numSpecs * sizeof(int)
  	        varSend: numMems*numReacts*numSpecs * sizeof(int)
  	        tgtVector: numMems*numReacts * sizeof(int)
  	        tgtMatrix: numMems*numReacts*numSpecs * sizeof(int)
  	        Cvector: numMems*numReacts * sizeof(float)
  	        Mfeed: numMems*numSpecs * sizeof(float)
  	        molSize: numMems*numSpecs * sizeof(float)
  	        membSize: numMems * sizeof(float)
  	        */
  	}

  	CPUfreeData();

  }

  float elapsedTimeMedium_ALLOC_UPLOAD = sdkGetAverageTimerValue(&timer_ALLOC_UPLOAD);
  float elapsedTimeMedium_FREE_DOWNLOAD = sdkGetAverageTimerValue(&timer_FREE_DOWNLOAD);
  float elapsedTimeMedium_CALC = sdkGetAverageTimerValue(&timer_CALC);
  timePerStepMedium_CALC /= numTrials;

  printf("\nnumMems = %u \nnumSims = %u \nnumTrials = %u", numMems, numSims, numTrials);

  printf("\nTotal GLOBAL memory occupied by simulations and system parameters: %u (vars) + %u (pars) = %u bytes \n", varSize, parSize, varSize + parSize);

  printf("\nelapsedTimeMedium_ALLOC_UPLOAD = %.3f(ms) \nelapsedTimeMedium_FREE_DOWNLOAD = %.3f(ms) \nelapsedTimeMedium_CALC = %.3f(ms)  \ntimePerStepMedium_CALC = %.3f(ms)", elapsedTimeMedium_ALLOC_UPLOAD, elapsedTimeMedium_FREE_DOWNLOAD, elapsedTimeMedium_CALC, timePerStepMedium_CALC);

  

  for (unsigned trial=0; trial<numTrials; trial++) {
  	printf("\n\nelapsedTime_CALC[%u] = %.3f(ms), total steps %u\n", trial, elapsedTime_CALC[trial], numStepsAllSims[trial]);
  	for (unsigned s=0; s<numSims; s++) printf("numSteps[%4u] = %u\n", s, numSteps[trial*numSims+s]);
  }

  sdkDeleteTimer(&timer_ALLOC_UPLOAD);
  sdkDeleteTimer(&timer_FREE_DOWNLOAD);
  sdkDeleteTimer(&timer_CALC);

  free(elapsedTime_CALC);
  free(numSteps);
  free(numStepsAllSims);

  fclose(stdout);

  hipDeviceReset();
  cudaCheckErrors("device reset");

  exit(EXIT_SUCCESS);
}
