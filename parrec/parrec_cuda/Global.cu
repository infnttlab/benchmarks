//-----------------------------------------------------------------------------
// Questo programma � stato creato da Rosa Brancaccio
//-----------------------------------------------------------------------------
// Dipartimento di Fisica
// Sezione Fisica Medica
// V.le Berti Pichat 6/2
// 40127 Bologna - ITALY
// 
// Tel.:   +39-051-2095132
// Tel.:   +39-051-2095134
// Fax.:   +39-051-2095047
// Email:  rossella_brancaccio@yahoo.it
// Web:    http://www.xraytomography.com
//-----------------------------------------------------------------------------
// GLOBAL.H
// mio file di intestazione con variabili globali, define, funzioni
//-----------------------------------------------------------------------------

//-----------------------------------------------------------------------------
// miei include 
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
#include "Global.h"
//-----------------------------------------------------------------------------
// variabili globali definite come extern nel global.h e qui inizializzate
//-----------------------------------------------------------------------------

//-----------------------------------------------------------------------------
// variabili globali definite come extern nel global.h e qui inizializzate
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
float	MAX_FLOAT = 999.0; //max_f;		// rossella rossella ?????? come li scelgo questi due vaolri??                                     		
float	MIN_FLOAT = 0.0; //min_f;		// non possono essere il minimo e il massimo dell'immagine xch� quando stretcho non funziona pi�   
//-----------------------------------------------------------------------------
struct 			data_tomo DataSet; 			// variabile globale per tutti i parametri in memoria relativi alle propriet� dell'immagine o sequenza
struct 			data_tomo DataSet2; 		// variabile globale per tutti i parametri in memoria relativi alla seconda sequenza
int 		   	current_bitmapID=0;  		// handle della bitmap da visualizzare
int 		   	preview_bitmapID=0;  		// handle della bitmap preview da visualizzare
unsigned char  *bits_preview2=NULL;  		// bits dell'immagine PREVIEW2 da visualizzare
int 		   	preview_bitmapID2=0;  		// handle della bitmap2 preview da visualizzare
int			   *Hist=NULL;	    			// puntatore dell'istogramma
int			   *Hist_height=NULL; 			// puntatore ai valori verticali dell'istogramma
int			   *Hist2=NULL;	    			// puntatore dell'istogramma
int			   *Hist2_height=NULL; 			// puntatore ai valori verticali dell'istogramma
unsigned char  *bits=NULL;  				// bits dell'immagine da visualizzare
unsigned char  *bits_preview=NULL;  		// bits dell'immagine PREVIEW da visualizzare
int 			*Led_Array=NULL;			// controlli dei led per i ranks
int				menubar_handle=0;			// handle del menubar
char			SCT_Data[MAX_STRING]={'\0'};// tutto il file SCT
int 			IMAGE1_LEFT=30;			
int 			IMAGE1_TOP=110;
int				*Min_array=NULL;				// array dei minimi di tutta la sequenza
int				*Max_array=NULL;				// array dei massimi di tutta la sequenza
float			*Minf_array=NULL;				// array dei minimi di tutta la sequenza
float			*Maxf_array=NULL;				// array dei massimi di tutta la sequenza
float			*xfslice_start=NULL;					// puntatore xstart alle coordinate del cerchio della slice
float			*xfslice_end=NULL;					// puntatore xend   alle coordinate del cerchio della slice

//-----------------------------------------------------------------------------
int status_ANT=STOP;
unsigned char	*bits_ant=NULL;
int *old_step=NULL;
int *new_step=NULL;
int *xant=NULL,*yant=NULL;

//-----------------------------------------------------------------------------
int	n_pxcenter=0;   // mi serve per dire che indice devo salvare quando trycenter � attivo
float *try_pxcenter=NULL;	// vettore dei punti di centri provati

