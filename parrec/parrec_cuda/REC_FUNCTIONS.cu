#include "hip/hip_runtime.h"
﻿//-----------------------------------------------------------------------------
// Questo programma è stato creato da Rosa Brancaccio
//-----------------------------------------------------------------------------
// Dipartimento di Fisica
// Sezione Fisica Medica
// V.le Berti Pichat 6/2
// 40127 Bologna - ITALY
// 
// Tel.:   +39-051-2095132
// Tel.:   +39-051-2095134
// Fax.:   +39-051-2095047
// Email:  rossella_brancaccio@yahoo.it
// Web:    http://www.xraytomography.com
//-----------------------------------------------------------------------------
// REC_FUNCTION.c
// mio file con le funzioni di ricostruzione per parallelizzazione
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
#include "Global.h"
#include <stdlib.h>
#include <hipfft/hipfft.h>

//-----------------------------------------------------------------------------
// variabili global per ricostruzione
//-----------------------------------------------------------------------------
float  *filter=NULL;						// FILTRO
float  *row_re=NULL,*row_im=NULL;
int  *x_start=NULL, *x_end=NULL;			// le due parti reale e immaginaria della riga
float   *slice=NULL,*slice_rank=NULL;		// le slice da allocare e ricostruire
float	*detector_weighing=NULL;			// il peso del detector
int		h_start=0,h_end=0;					// la suddivisione degli angoli in ranks
int		*d_x_start=NULL, *d_x_end=NULL;			// vettori con i punti x di partenza e arrivo in funzione di y
float	*row_modified=NULL;					// riga modificata per halfscan si allocherà solo se servirà
//-----------------------------------------------------------------------------
// per allocare row e filter è necessario calcolare la lunghezza degli array nel dominio di fourier
// è necessario avere almeno il doppio meno uno della dimensione dei punti da elaborare
// inoltre se si usa un array con lunghezza potenza di 2 la FFT lavora meglio per cui:
//-----------------------------------------------------------------------------


//--- dichiarazioni kernel ----//
__global__ void zeropaddingslice(float*, int);
__global__ void creation_xStartEnd(int*, int*, float, float, float, int, int, int, int, int, int);
__global__ void zp_d_signal(hipfftComplex*, int);
__global__ void initialize_signal(hipfftComplex*, float*, int , int , int , int, float*);
__global__ void convolution(hipfftComplex*, float*, int);
__global__ void geometry_reconstruction(hipfftComplex*, float*, float*, int*, int*, float, float, float, float, float, float, float, float, float, float, float, float, float, float, int, int, int, int, int, int, int, int, int, int,int,float,float,float,float,float,float,float,float,float,float,float,float,float,float,int);
//-------------------------------


int CalcTheMinimumNumberPowerOfTwoAfter(int dim) {

	int	N;
	float N_float;

	// si calcola il logaritmo in base e (naturale)
	N_float = log( (float)dim );

	// si trasforma in base 2
	N_float = N_float / log(2.);

	// si somma uno per arrotondare in eccesso (anche se venisse 10.2 vorremmo usare 11 perchè 10 non basterebbe!)
	N_float += 1.;

	// N è l'intero della potenza di due dell'intero di N_float
	N_float = (float) ( (int)N_float );
	N = (int) ( pow(2.,N_float) );

	return N;
}
//-----------------------------------------------------------------------------
// pesatura per geometria di scanning: nel fan beam la pesatura dei punti del 
// rivelatore va fatta in funzione della geometria e si distingue in 
// rivelatori con raggi equispatiati o equiangolari (vedere il Kak)
// ora calcola il peso anche per feldkamp!!! la differenza sta nel fatto che dw diventa 2D
//-----------------------------------------------------------------------------
int CalculateWeighing_OnTheBaseOfCTGeometry(int detector_geometry,int FFT_start,int nrays,float sod,float pxsize,float pxcenter) {

	switch(detector_geometry) {

		// pesatura per equispaziati SOLO FAN BEAM ---W1--- N.B. non dipende da h quindi facciamo i calcoli fuori dal ciclo!!!
		// tutti gli array dei valori che vanno nel dominio delle frequenze partono da 0 a FFT_Lenght
		// ma noi abbiamo dei valori del detector a partire da FFT_start
		// quindi k è l'indice degli array nel dominio delle frequenze e va da zero a FFT_lenght
		// mentre i è l'indice dei valori del detector e si trova a i=k-FFT_start e va da 0 a width-1
		// n è l'indice definito in modo che sia zero in pxcenter n=k-FFT_start-pxcenter
		// n va da -pxcenter a width - pxcenter ed è 0 in i=pxcenter e k=FFT_start+pxcenter
		// pesatura per equispaziati SOLO FAN BEAM è il valore di sod /sqrt(sod*sod+n*n*pxsize*pxsize) perchè EQUISPAZIATI 
		case(EQUISPACED_RAYS) : {

			double sod_quadro, pxsize_quadro,n;
			int k,width;

			width=DataSet.Width;
			sod_quadro = (double)sod*(double)sod;   
			pxsize_quadro = (double)pxsize*(double)pxsize;
			for(k=FFT_start; k<FFT_start+width; k++) {  // qui ho messo width al posto di nrays per halfscan
				n = (double)k-(double)FFT_start-(double)pxcenter; // l'ho trasformato in double
				detector_weighing[k-FFT_start] = sod / ( sqrt(sod_quadro+n*n*pxsize_quadro));
				// diminuiamo la memoria da allocare (dato che si tratta di un double) allocando width (e non FFT_lenght) e spostando poi k da FFT_start
			}
			return 0;
		}

		case(EQUIANGULAR_RAYS) : {
			
			// da sviluppare
			return 0;
		}

		// in questo caso il detector weighing è bidimensionale, dipende da x e da y del detector in funzione di pxcenter e pzcenter
		// ma non è dipendente da teta (angolo di rotazione) perchè sorgente e rivelatore si muovono solidalmente
		case(FELDKAMP_CONEBEAM) : {
			
			float	sod_f=(float)sod,pzcenter=DataSet.pzcenter,pysize=DataSet.pysize;
			float	sod_fq,kfq,jfq;
			int		j,k;
			int		width=DataSet.Width,height=DataSet.Height;

			sod_fq = sod_f*sod_f; // ci serve per Feldkamp weight

			for(j=0; j<height; j++) {
				// se si filtra per Feldkamp cambia il peso e va applicato nel ciclo perchè oltre che da x dipende anche da y
				jfq = (float)(j)-pzcenter;
				jfq = jfq*pysize;
				jfq = jfq*jfq;
				for(k=FFT_start; k<FFT_start+width; k++) {
					kfq = (float)(k)-(float)FFT_start-pxcenter;
					kfq = kfq*pxsize;
					kfq = kfq*kfq;
					detector_weighing[j*width+k-FFT_start] = (sod_f/sqrt(sod_fq+kfq+jfq));
					// diminuiamo la memoria da allocare (dato che si tratta di un double) allocando width (e non FFT_lenght) e spostando poi k da FFT_start
				}
			}

			return 0;
		}

	}

	return 0;
}
//-----------------------------------------------------------------------------
// calcolo del filtro da applicare nello spazio di Fourier a partire da RAMP e con BUTTER
// bisogna passargli il tipo di W2 che vogliamo applicare
//-----------------------------------------------------------------------------
int CalculateFourierFilter(int FFT_lenght, int FFT_start, double W2) {

	int k;
	double cutoff,delta_f,freq,butter;

	// inizializzazione: zero-padding
	for(k=0; k<FFT_lenght; k++) 
		filter[k]=0.;

	// in TEORIA: si aumenta linearmente il valore del filtro a partire dal centro in cui è zero // cutoff è per definizione 0.5
	// in realtà poichè la funzione FFT trasforma nello spazio di fourier con la frequenza di nyquist all'esterno e non al centro
	// invertiamo la definizione del filtro piuttosto che invertire due volte la projection nello spazio di fouirer
	// per tutti i dettagli vedi mio quaderno, il filtro qui implementato è 0.5 al centro (dove c'è Nyquist) e va a zero ai bordi

	// filtro di tipo ramp 
	cutoff = 0.5*W2;   // il filtro deve andare a 1/2pxsize in più c'è W2 che però ho messo a 1. altrimenti diventava pesantissimo!!!
	delta_f = 1.0 / (float)FFT_lenght;
	freq = 0.0;
	filter[0] = 0.;
	filter[FFT_lenght-1] = 0.;
	filter[FFT_start] = cutoff;

	// riempiamo il filtro di ramp
	for (k=1; k<=(FFT_start); k++) {  
		freq += delta_f;
		if (freq <= cutoff)										
			filter[k] = filter[FFT_lenght-k] = (double)k *delta_f*W2; // questo peso ci serve perchè il filtro vada anzichè a 0.5 a 0.5*1/pxsize
		else
			filter[k] = filter[FFT_lenght-k] = 0.0;
	}

	// filtro butter
	freq=0.;
	for (k=1; k<=(FFT_start); k++) {  
		freq += delta_f;
	    butter = 1.0 / sqrt(1.0 + pow((double)(freq/cutoff),10.));
	    if (butter<=0.0)
	       filter[k] = filter[FFT_lenght-k] = 0.0;
	    else
	       filter[k] = filter[FFT_lenght-k] = filter[k] * butter;
	}

	return 0;
}
//---------------------------------------------------------
// calcolo del peso per equispaziati U --- W3 ---
// questa funzione alloca l'array Uquadro_teta_zero e lo calcola
//---------------------------------------------------------
int	CalcolateWeighing_EquispacedRays(float sod,float rxsize) {

	int		y=0,width=DataSet.Width;
	float	val=0.,U,Uquadro;
	float	l;
	float	*Uquadro_teta_zero=NULL;			// il peso Uquadro

	// si alloca U
	free(Uquadro_teta_zero);		Uquadro_teta_zero = NULL;		Uquadro_teta_zero= (float *) malloc ( sizeof(float)*width+1);

	// si calcola l
	l = rxsize*((float)width/2.);

	// si calcola U di teta=0
	for(y=0; y<width; y++) {
		val = sod -( (float)y*rxsize-l);
		U = sod/val;
		Uquadro = U * U;
		Uquadro_teta_zero[y] = Uquadro;
	}

	printf("\n y\tU quadro di teta zero");
	for(y=0; y<width; y++) 
		printf("\n%d\t%f",y,Uquadro_teta_zero[y]);
	fflush(0);

	free(Uquadro_teta_zero);		Uquadro_teta_zero = NULL;	

	return 0;
}
//-----------------------------------------------------------------------------
// funzione per retroproiezione filtrata CON constrain circle SOLO MATEMATICA
// N.B. PRIMA DI ACCEDERE A QUESTA FUNZIONE BISOGNA ALLOCARE SLICE ROW RE ROW IM E FILTER
// INOLTRE è NECESSARIO AVER CARICATO IN MEMORIA IMAGE DA ELABORARE E AVER CALCOLATO FILTER E PESI
// // **** 2013 07 19 ANCHE LOCAL CT **** SENZA HALFSCAN
//-----------------------------------------------------------------------------
int BackProjectFilteredDataOnSlice(int FFT_lenght,int FFT_start,float xs,float ys,float	xd0,float yd0,float	xd1,float yd1,int numBlocks,int threadsPerBlock,int dimBlock) {

	// indici
	int		h,riga;
	// dati da SCT
	int		nrays,width;
	float	arange,nangles;
	float	pxsize,rxsize,sod;
	float	wmezzif;
	// variabili per stare dentro al cerchio e localtomography
	float	r=0.,fanangle=0.,l=0.;
	float	r_pixel=0.;
	int		y_start=0,y_end=0;
	// variabili per rotazione
	float	teta,teta_rad,teta_step,sin_teta,cos_teta;

	float	xs_rot,ys_rot;
	float	xd0_rot,yd0_rot,xd1_rot,yd1_rot;
	// variabili per rette
	float	A,B;						// coefficienti della retta del detector ruotato (da D0 a D1)
	float	xd_min,xd_max,yd_min,yd_max; // ci servono per vedere se c'è l'intersezione nel range del detector
	// variabili per interpolazione
	float	xs_rot_quadro,ys_rot_quadro;	// sull'asse che va dalla sorgente all'origine
	//float	val_pesato_U;
	float	sod_quadro;
// per salvare slice parziale
//	char	path[MAX_STRING],path_SDT[MAX_STRING],path_SPR[MAX_STRING],txt1[SHORT2_STRING];
//	char	txt[MEDIUM_STRING];

	// dati SCT
	width = DataSet.Width;
	wmezzif = ((float)width)/2.;   // width mezzif è width trasformato in float e diviso 2
	nrays	= DataSet.nrays;		// vale width a meno che non si faccia mezzo giro HALFSCAN
	sod		= DataSet.sod;
	pxsize	= DataSet.pxsize;					
	rxsize	= DataSet.rxsize;

	// N.B. nel mio sistema di riferimento la sorgente è in alto, il rivelatore è in basso
	// in questo modo sono consistente con me stessa ma rispetto a imgrec ho dovuto inserire il meno davanti a arange
	arange =  - DataSet.arange; 
	nangles = (float) DataSet.nangles;
	teta_step = arange / nangles; // teta step deve essere negativo altrimenti non cambierebbe il verso di rotazione

	// calcoliamo r e l in funzione di width e non nrays perchè nrays potrebbe essere metà
	l = wmezzif*rxsize; // l è la metà del lato del quadrato della slice in mm

	// calcoliamo fanangle e r con - shift 
	Calculate_r_fi(FAN_BEAM,&r,&fanangle);
	r_pixel = r/rxsize;

	// calcolo di sod quadro per pesatura su U
	sod_quadro = sod*sod;

	//host:
	int dominio = width*width;
	//device:
	float *dev_slice;

	hipMalloc((void**)&dev_slice, (dominio * sizeof(float)+1));//all'indirizzo di dev_slice alloco uno spazio di
																//	w*w*(dim di 1float su quella macchina in bytes)
	printf("\nnumBlocks = %d   threadsPerBlock = %d   dimBlock = %d\n",numBlocks,threadsPerBlock,dimBlock);

	zeropaddingslice<<<numBlocks, threadsPerBlock>>>(dev_slice, dominio);

// creazione di y_start/end e x_start/end[y]
	if(DataSet.localCT_status == FALSE ) {
		y_start = (int)(wmezzif - r_pixel)+CUTSLICE;
		y_end   = (int)(wmezzif + r_pixel)-CUTSLICE;
		// controllo y_start e y_end nei limiti
		y_start = (y_start >= 0    ) ? y_start : 0 ;
		y_start = (y_start < width ) ? y_start : width-1 ;
		y_end   = (y_end >= 0    ) ? y_end : 0 ;
		y_end   = (y_end < width ) ? y_end : width-1 ;
	} else {
		// localtomography
		y_start = DataSet.localCT_y0;
		y_end   = DataSet.localCT_y1;
	}

	// x_start/end:
	int localCT_status = DataSet.localCT_status;
	int localCT_x0 = DataSet.localCT_x0;
	int localCT_x1 = DataSet.localCT_x1;

	//device:
	// per stare dentro al cerchio r
	if(!d_x_start)
		hipFree(d_x_start);
	d_x_start = NULL;
	if(!d_x_end)
		hipFree(d_x_end);
	d_x_end = NULL;

	hipMalloc((void**)&d_x_start, (width * sizeof(int)+1));
	hipMalloc((void**)&d_x_end, (width * sizeof(int)+1));
	
	creation_xStartEnd<<<numBlocks, threadsPerBlock>>>(d_x_start, d_x_end, r, l, rxsize, localCT_status, localCT_x0, localCT_x1, width, y_start, y_end);
	

//---------------------------------------------------------
// ciclo su teta in funzione di h
//---------------------------------------------------------

	for(h=0; h<(int)nangles; h++) {
		teta = (float)h*teta_step;

		// teta è in gradi x trasformarlo in radianti si deve applicare la	teta rad = 2 pigreco * teta grad / 360.
		teta_rad = 2.* PIGRECO * teta / 360.;

		hipfftComplex *d_signal;
		hipMalloc((void**)&d_signal, sizeof(hipfftComplex)*(FFT_lenght+1));

		zp_d_signal<<<numBlocks, threadsPerBlock>>>(d_signal,FFT_lenght);

		int dimImageFL = DataSet.Width*DataSet.Height;
		riga=h*width;

		float *d_Image_FL;
		hipMalloc((void**)&d_Image_FL, sizeof(float)*(dimImageFL+1));
		hipMemcpy(d_Image_FL, DataSet.Image_FL, sizeof(float)*(dimImageFL+1), hipMemcpyHostToDevice);

		float *d_detector_weighing;
		hipMalloc((void**)&d_detector_weighing, sizeof(float)*(width+1));
		hipMemcpy(d_detector_weighing, detector_weighing, sizeof(float)*(width+1), hipMemcpyHostToDevice);

		hipfftComplex *d_signal_image;
		hipMalloc((void**)&d_signal_image, sizeof(hipfftComplex)*(FFT_lenght+1));

		initialize_signal<<<numBlocks, threadsPerBlock>>>(d_signal, d_Image_FL,FFT_start, width, riga, FFT_lenght, d_detector_weighing);

		hipFree(d_detector_weighing);

		//se la dimensione dell'array row_re (e row_im) è minore di 2^13 allora si fa l'fft del NR, altrimenti la cuFFT
		if (FFT_lenght<pow(2,13)){
			
			hipfftComplex *h_signal = (hipfftComplex *) malloc(sizeof(hipfftComplex)*(FFT_lenght+1));
			hipMemcpy(h_signal, d_signal,sizeof(hipfftComplex)*(FFT_lenght+1), hipMemcpyDeviceToHost);
			
			for(int ii=0; ii<FFT_lenght; ii++){
				row_re[ii] = h_signal[ii].x;
				row_im[ii] = h_signal[ii].y;
			}
			// fast fourier transform
			FFT(row_re,row_im,FFT_lenght);

			// convoluzione con il filtro 
			for(int k=0; k<FFT_lenght; k++){
				row_re[k]=row_re[k]*filter[k];
				row_im[k]=row_im[k]*filter[k];
			}
				
			// inverse fft: inverse fast fourier transform
			InvFFT(row_re,row_im,FFT_lenght);
			
			for(int ii=0; ii<FFT_lenght; ii++){
				h_signal[ii].x = row_re[ii];
				h_signal[ii].y = row_im[ii];
			}
			
			hipMemcpy(d_signal, h_signal, sizeof(hipfftComplex)*(FFT_lenght+1), hipMemcpyHostToDevice);
			hipFree(h_signal);
		}
		else {
			
			//mi preparo i piano per Fourier:
			hipfftHandle plan;
			hipfftPlan1d(&plan, FFT_lenght, HIPFFT_C2C, 1);
	
			// Transformo
			hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);
	
			float *d_filter;
			hipMalloc((void**)&d_filter, (FFT_lenght+1) * sizeof(float));
			hipMemcpy(d_filter, filter, (FFT_lenght+1) * sizeof(float), hipMemcpyHostToDevice);
	
			//Convoluzione
			convolution<<<numBlocks, threadsPerBlock>>>(d_signal, d_filter, FFT_lenght);
	
			hipFree(d_filter);
	
			//Antitrasformo
			hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
	
			hipfftDestroy(plan);
		}

		// calcolo di sin teta e cos teta per non ripeterlo ogni volta
		sin_teta = sin(teta_rad);
		cos_teta = cos(teta_rad);

		// coordinate ruotate della sorgente rispetto all'origine 0,0
		xs_rot = xs*cos_teta-ys*sin_teta;
		ys_rot = xs*sin_teta+ys*cos_teta;

		// coordinate ruotate della sorgente al quadrato: ci serve per calcolare W3, le calcoliamo qui perchè non dipendono da i
		xs_rot_quadro = xs_rot*xs_rot;
		ys_rot_quadro = ys_rot*ys_rot;

		// calcoliamo d0 e d1 rot per tracciare le rette e interpolare sulla slice
		xd0_rot = xd0*cos_teta-yd0*sin_teta;
		yd0_rot = xd0*sin_teta+yd0*cos_teta;
		xd1_rot = xd1*cos_teta-yd1*sin_teta;
		yd1_rot = xd1*sin_teta+yd1*cos_teta;


		xd_min = ( xd0_rot < xd1_rot ) ? xd0_rot : xd1_rot;
		xd_max = ( xd0_rot > xd1_rot ) ? xd0_rot : xd1_rot;
		yd_min = ( yd0_rot < yd1_rot ) ? yd0_rot : yd1_rot;
		yd_max = ( yd0_rot > yd1_rot ) ? yd0_rot : yd1_rot;

		// traccaire le rette:
		int caso = 0;

		// Caso detector non verticale: si tracciano le rette
		if( xd0_rot != xd1_rot ) {

			A = (yd0_rot-yd1_rot)/(xd0_rot-xd1_rot);
			B = yd0_rot - xd0_rot*A;

			caso = 1;
		  		}
		// Caso detector verticale: si tracciano le rette
		else{
			caso = 2;
			A = 0.;
			B = 0.;
		}

		//ora mi calcolo il lato minimo del quardrato entro cui deve essere inscritto il cerchio di ricostruzione (cioè il diametro del cerchio ovviamente):
		int side_square_circle = (y_end+1)-y_start; // visto che è il diametro potrebbe essere anche r_pixel*2--> controllare che valga per tutti i casi!!(localtomography)

		int dimgrid = (side_square_circle+(dimBlock-1))/dimBlock;

		dim3 sizeblock(dimBlock,dimBlock);
		dim3 sizegrid(dimgrid,dimgrid);

		geometry_reconstruction<<<sizegrid, sizeblock>>>(d_signal, dev_slice, d_Image_FL, d_x_start, d_x_end, sod_quadro, pxsize, r, rxsize, xs, ys, teta_rad, xd0, xd1, yd0, yd1, A, B, l, caso, nrays, FFT_lenght, FFT_start, h, localCT_status, localCT_x0, localCT_x1, width, y_start, y_end,sin_teta,cos_teta, xs_rot, ys_rot, xd0_rot, xd1_rot, yd0_rot,yd1_rot, xd_min,xd_max,yd_min, yd_max, xs_rot_quadro,  ys_rot_quadro,riga);

		hipMemcpy(slice, dev_slice, (dominio * sizeof(float)+1), hipMemcpyDeviceToHost);
		
		hipFree(d_Image_FL);
		hipFree(d_signal);

	} // fine di teta in funzione di h

	hipFree(dev_slice);
	hipFree(d_x_start);
	hipFree(d_x_end);

	return 0;
}

//-----------------------------------------------------------------------------
// calcolo del peso --- W3 --- vale 1/Uquadro per equispaziati
//-----------------------------------------------------------------------------
int CalculateWeight3(int detector_geometry,float l,double sod_quadro,double xs_rot_quadro,double ys_rot_quadro,float xs_rot,float ys_rot,float *W3_U) {

	int		x,y;
	int		width=DataSet.Width;
	float	xf,yf;
	float	rxsize=DataSet.rxsize;
	float	U,U_quadro,uno_su_U_quadro;

	//printf("\n%s\n",DataSet.Name);
	
	for(y=0; y<width; y++) {

		// calcolo di yf
		yf = (float)y*rxsize -l;	// qui andrebbe + rxsize/2

		//printf("\n");

		// ciclo su x
		for(x=0; x<width; x++) {

			// calcolo di xf
			xf = (float)x*rxsize -l;		// qui andrebbe + rxsize/2

			// peso per la distanza dalla proiezione sulla retta della sorgente ---W3--- EQUISPAZIATI
			U = (sod_quadro+xs_rot_quadro+ys_rot_quadro-2.*xs_rot*xf-2.*ys_rot*yf)/sod_quadro;  
			U_quadro = U * U; 
			uno_su_U_quadro = 1. / U_quadro;

			W3_U[y*width+x]=uno_su_U_quadro ;
			//printf("%f\t",uno_su_U_quadro);
		}
	}


	return 0;
}

//---------------------------------------------------------
// unica funzione che in base a DataSet calcola il raggio r e fan angle
// calcoliamo fanangle e r con - shift 
//---------------------------------------------------------
void	Calculate_r_fi(int geometry,float *r,float *fi) {

	float	fanangle,ray;
	float	wmezzif,pxsize,sdd,odd,sod;
	float	pxcenter;
	int		width;

	// dati da DataSet
	width = DataSet.Width;
	wmezzif = (float)width/2.;
	pxsize = DataSet.pxsize;
	sdd = DataSet.sdd;
	odd = DataSet.odd;
	sod = DataSet.sod;
	pxcenter = DataSet.pxcenter;

	if( geometry == FAN_BEAM ) {

		// calcolo dell'angolo di fan
		fanangle = atanf( wmezzif*pxsize/sdd );  // radianti
		ray = ((pxsize*((float)width-pxcenter)/(tanf(fanangle)))-odd) * sin(fanangle);

	} else if ( geometry == CONE_BEAM ) {

		// ---------
		// FELDKAMP
		// ---------
		fanangle = atanf( wmezzif*pxsize/sdd );  // radianti
		ray = sod * sinf(fanangle);

	} else { // PARALLEL BEAM

		fanangle =-99.;
		ray = -99.;
		return;
	}

	// in imgrec a destra r diminuisce a sinistra r aumenta
	// immisioni dati calcolati
	*r = ray;
	*fi = fanangle;

	return;

}
//-----------------------------------------------------------------------------
//  valore assoluto di un float
//-----------------------------------------------------------------------------
float Absolute(float val) {

	if( val >= 0. )
		return val;
	else
		return -val;
}

//-----------------------------------------------------------------------------
//	si compone pathname dai dati inseriti in data set
// se n=-1 allora dark se n=-2 allora bak
//-----------------------------------------------------------------------------
void	ComposeFileNameFromDataSet(char *PathName) {
		
	char	txt[MAX_STRING],txt1[SHORT_STRING];
	
	txt[0]='\0';
	txt1[0]='\0';
	
	sprintf(txt,DataSet.Path);
	strcat(txt,DataSet.Name);  // qualsiasi sia il nome della sequenza qui viene immesso

	// se è una proiezione ci può essere una dark o una izero
	if( DataSet.TomographicType == PROJECTION ) {
		
		if(DataSet.current_image==DARK) {
			sprintf(txt1,"drk");
			strcat(txt,txt1); 	
		} else if(DataSet.current_image==IZERO) {
			sprintf(txt1,"bak");
			strcat(txt,txt1); 	
		} else if( DataSet.current_image != NO_SEQ ) {
			sprintf(txt1,"_%d",DataSet.current_image);
			strcat(txt,txt1); 	
		}

	// non è una proiezione ma è riconsciuta come parte di una sequenza con un numero
	} else if( DataSet.current_image != NO_SEQ ) {
		sprintf(txt1,"_%d",DataSet.current_image);
		strcat(txt,txt1);
	} 
		
	if( DataSet.File_Type == SDT)
		strcat(txt,".sdt");
	else if( DataSet.File_Type == JPG)
		strcat(txt,".jpg");
	else if( DataSet.File_Type == BMP)
		strcat(txt,".bmp");
	else if( DataSet.File_Type == TIF)
		strcat(txt,".tif");

	CopyStringToStringAndTerminateIt(PathName,txt);
	
	return;
	
	
}
//-----------------------------------------------------------------------------
// copia string_tocopy in string_target e poi la termina
//-----------------------------------------------------------------------------
void CopyStringToStringAndTerminateIt(char *string_target,char *string_tocopy) {

	int k=0,len=0;
	
	len = (int) strlen(string_tocopy);
	for(k=0; k<len; k++)
		string_target[k]=string_tocopy[k];
	string_target[len]='\0';

	return;
}
//-----------------------------------------------------------------------------
// si calcolano max e min dell'immagine corrente in base al dataset
//-----------------------------------------------------------------------------
void MaxMin(int dataset) {
	
	int x=0, y=0;
	int min=MAX_16BIT,max=ZERO,val=0;
	float min_f=FLT_MAX,max_f=-FLT_MIN,val_f=0;
	int		width,height;
	
	if(dataset == DATASET1) {

		width=DataSet.Width;
		height=DataSet.Height;

		if(DataSet.Data_Type == U16 ) {
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val=(int)DataSet.Image_U16[y*width+x];
					min = ( min < val ) ? min : val;
					max = ( max > val ) ? max : val;
				}
			}
			DataSet.min = min;
			DataSet.max = max;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			if( DataSet.Number_of_Images > SINGLE_IMAGE && Min_array != NULL && Max_array != NULL ) {
				Min_array[DataSet.current_image-DataSet.Seq_start]=min;
				Max_array[DataSet.current_image-DataSet.Seq_start]=max;
			}

		}
	
		if(DataSet.Data_Type == U8) {
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val=(int)DataSet.Image_U8[y*width+x];
					min = ( min < val ) ? min : val;
					max = ( max > val ) ? max : val;
				}
			}
			DataSet.min = min;
			DataSet.max = max;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			if( DataSet.Number_of_Images > SINGLE_IMAGE && Min_array != NULL && Max_array != NULL ) {
				Min_array[DataSet.current_image-DataSet.Seq_start]=min;
				Max_array[DataSet.current_image-DataSet.Seq_start]=max;
			}

		}
		
		if(DataSet.Data_Type == FLT) {	   // rossella float
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val_f=DataSet.Image_FL[y*width+x];
					min_f = ( min_f < val_f ) ? min_f : val_f;
					max_f = ( max_f > val_f ) ? max_f : val_f;
				}
			}
			DataSet.min_f = min_f;
			DataSet.max_f = max_f;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			if( DataSet.Number_of_Images > SINGLE_IMAGE && Minf_array != NULL && Maxf_array != NULL ) {
				Minf_array[DataSet.current_image-DataSet.Seq_start]=min_f;
				Maxf_array[DataSet.current_image-DataSet.Seq_start]=max_f;
			}
		
		}
	// fine di DATSET1
	} else {

		width=DataSet2.Width;
		height=DataSet2.Height;

		if(DataSet2.Data_Type == U16 ) {
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val=(int)DataSet2.Image_U16[y*width+x];
					min = ( min < val ) ? min : val;
					max = ( max > val ) ? max : val;
				}
			}
			DataSet2.min = min;
			DataSet2.max = max;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			//if( DataSet2.Number_of_Images > SINGLE_IMAGE && Min_array != NULL && Max_array != NULL ) {
			//	Min_array[DataSet2.current_image-DataSet2.Seq_start]=min;
			//	Max_array[DataSet2.current_image-DataSet2.Seq_start]=max;
			//}

		}
	
		if(DataSet2.Data_Type == U8) {
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val=(int)DataSet2.Image_U8[y*width+x];
					min = ( min < val ) ? min : val;
					max = ( max > val ) ? max : val;
				}
			}
			DataSet2.min = min;
			DataSet2.max = max;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			//if( DataSet2.Number_of_Images > SINGLE_IMAGE && Min_array != NULL && Max_array != NULL ) {
			//	Min_array[DataSet2.current_image-DataSet2.Seq_start]=min;
			//	Max_array[DataSet2.current_image-DataSet2.Seq_start]=max;
			//}

		}
		
		if(DataSet2.Data_Type == FLT) {	   // rossella float
		
			for(x=0; x<width; x++) {
				for(y=0; y<height; y++) {
					val_f=DataSet2.Image_FL[y*width+x];
					min_f = ( min_f < val_f ) ? min_f : val_f;
					max_f = ( max_f > val_f ) ? max_f : val_f;
				}
			}
			DataSet2.min_f = min_f;
			DataSet2.max_f = max_f;
			// se c'è una sequenza vale la pena inserire i dati calcolati in max min
			//if( DataSet2.Number_of_Images > SINGLE_IMAGE && Minf_array != NULL && Maxf_array != NULL ) {
			//	Minf_array[DataSet2.current_image-DataSet2.Seq_start]=min_f;
			//	Maxf_array[DataSet2.current_image-DataSet2.Seq_start]=max_f;
			//}
		
		}
	} // fine di dataset2
	return;
}

//-----------------------------------------------------------------------------
// si salva l'immagine in input come SDT nel path specificato e con le dimensioni specificate 
// ci sono due funzioni: questa è per i float
//-----------------------------------------------------------------------------
int Save_SDT_GenericFile_FLT(float *image,char *path,int width,int height) {
	
	FILE 	*f;
	//size_t	err=0;
	size_t	count_byte=0;
	//ssize_t		size=0;
	//int		fsize=0;


	// il nome del file da salvare si trova in path
	// si salva il file SDT nel percorso stabilito

	// si scrive il file
	f = fopen(path,"wb");
	count_byte = sizeof(float)*width*height;
	fwrite(image,count_byte,1,f); 
	fclose(f);
	
	return 0;
}
//-----------------------------------------------------------------------------
// si salva l'SPR nel path specificato e con le dimensioni specificate 
// e con i paramteri specificati
// **** size_policy: SI SALVANO ANCHE PSIZE O RSIZE SE COSì SI VUOLE *************
//-----------------------------------------------------------------------------
int Save_SPR_GenericFile(char *path,int width,int height,int IMG_TYPE,int DATA_TYPE,int size_policy) {
	

	FILE 	*f;
	
	// Save spr file
	f = fopen(path, "w");
	if (!f)
	{
		printf("\nWarning! Unable to open spr selected file");
		return -1;
	}

	// si fa l'spr
	fprintf(f,"%d\n",IMG_TYPE);
	fprintf(f,"%d\n",width);
	fprintf(f,"0.000000\n");
	fprintf(f,"0.000000\n");
	fprintf(f,"%d\n",height);
	if( size_policy == SAVE_PSIZE ) {
		fprintf(f,"%.6f\n",DataSet.pxsize);
		fprintf(f,"%.6f\n",DataSet.pysize);
	} else if( size_policy == SAVE_RSIZE ) {
		fprintf(f,"%.6f\n",DataSet.rxsize);
		fprintf(f,"%.6f\n",DataSet.rysize);
	} else {
		fprintf(f,"0.000000\n");
		fprintf(f,"0.000000\n");
	}
	fprintf(f,"%d\n",DATA_TYPE);
	fclose(f);	
	
	return 0;
}
//-----------------------------------------------------------------------------
// FFT by numerical recipes in c
// Replaces data[1..2*nn] by its discrete Fourier transform, if isign is input as 1; or replaces
// data[1..2*nn] by nn times its inverse discrete Fourier transform, if isign is input as −1.
// data is a complex array of length nn or, equivalently, a real array of length 2*nn. nn MUST
// be an integer power of 2 (this is not checked for!).
//-----------------------------------------------------------------------------
void four1(float data[], unsigned long nn, int isign) {

	unsigned long n,mmax,m,j,istep,i;
	double wtemp,wr,wpr,wpi,wi,theta;	//Double precision for the trigonomet
	float tempr,tempi;					//ric recurrences.

	n=nn << 1;
	j=1;
	for (i=1; i<n; i+=2) {		//This is the bit-reversal section of the //routine.
		if (j > i) { 
			SWAP(data[j],data[i]); //Exchange the two complex numbers.
			SWAP(data[j+1],data[i+1]);
		}
		m=nn;
		while (m >= 2 && j > m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	}

	// Here begins the Danielson-Lanczos section of the routine.
	mmax=2;
	while (n > mmax) {			//Outer loop executed log2 nn times.
		istep=mmax << 1;
		theta=isign*(6.28318530717959/mmax); //Initialize the trigonometric recurrence.
		wtemp=sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi=sin(theta);
		wr=1.0;
		wi=0.0;
		for (m=1;m<mmax;m+=2) {			//Here are the two nested inner loops.
			for (i=m;i<=n;i+=istep) {
				j=i+mmax;				//This is the Danielson-Lanczos formula:
				tempr=wr*data[j]-wi*data[j+1]; 
				tempi=wr*data[j+1]+wi*data[j];
				data[j]=data[i]-tempr;
				data[j+1]=data[i+1]-tempi;
				data[i] += tempr;
				data[i+1] += tempi;
			}
			wr=(wtemp=wr)*wpr-wi*wpi+wr; //Trigonometric recurrence.
			wi=wi*wpr+wtemp*wpi+wi;
		}
		mmax=istep;
	}
	return;
}
//-----------------------------------------------------------------------------
// fast fourier transform
//-----------------------------------------------------------------------------
void	FFT(float *row_re,float *row_im,int FFT_lenght) {
	
	float	*data=NULL;
	unsigned long nn;
	int		k;

	nn = FFT_lenght;
	data = (float *) malloc ( sizeof(float)*2*nn+1);

	for(k=0; k<FFT_lenght; k++ ) {
		// parte reale 1, 3 ... numeri dispari
		data[2*k+1]=row_re[k];
		// parte immaginaria 2,4...numeri pari a partire dal 2
		data[2*k+2]=row_im[k];
	}

	// FFT by numerical recipes in c
	// Replaces data[1..2*nn] by its discrete Fourier transform, if isign is input as 1; or replaces
	// data[1..2*nn] by nn times its inverse discrete Fourier transform, if isign is input as −1.
	// data is a complex array of length nn or, equivalently, a real array of length 2*nn. nn MUST
	// be an integer power of 2 (this is not checked for!).
	four1(data,nn,1);

	for(k=0; k<FFT_lenght; k++ ) {
		// parte reale 1, 3 ... numeri dispari
		row_re[k]=data[2*k+1];
		// parte immaginaria 2,4...numeri pari a partire dal 2
		row_im[k]=data[2*k+2];
	}


	free(data);

	return;
}
//-----------------------------------------------------------------------------
// inverse fft: inverse fast fourier transform
//-----------------------------------------------------------------------------
void	InvFFT(float *row_re,float *row_im,int FFT_lenght) {
	
	float	*data=NULL;
	unsigned long nn;
	int		k;

	nn = FFT_lenght;
	data = (float *) malloc ( sizeof(float)*2*nn+1);

	for(k=0; k<FFT_lenght; k++ ) {
		// parte reale 1, 3 ... numeri dispari
		data[2*k+1]=row_re[k];
		// parte immaginaria 2,4...numeri pari a partire dal 2
		data[2*k+2]=row_im[k];
	}

	// FFT by numerical recipes in c
	// Replaces data[1..2*nn] by its discrete Fourier transform, if isign is input as 1; or replaces
	// data[1..2*nn] by nn times its inverse discrete Fourier transform, if isign is input as −1.
	// data is a complex array of length nn or, equivalently, a real array of length 2*nn. nn MUST
	// be an integer power of 2 (this is not checked for!).
	four1(data,nn,-1);

	for(k=0; k<FFT_lenght; k++ ) {
		// parte reale 1, 3 ... numeri dispari
		row_re[k]=data[2*k+1];
		// parte immaginaria 2,4...numeri pari a partire dal 2
		row_im[k]=data[2*k+2];
	}

	free(data);

	return;
}
//-----------------------------------------------------------------------------
// Stampa del dataset scelto nello standard output
//-----------------------------------------------------------------------------
void PrintData_DataSet(void) {

		
		printf("\n-----------------------------------------------------------------------------"); fflush(0);         			
		printf("\n-----------PRINTING-OF-DATASET1----------------------------------------------"); fflush(0);         			
		printf("\n-----------------------------------------------------------------------------"); fflush(0);         			
		printf("\nDataSet.Number_of_Images 		 	 = %d  ",DataSet.Number_of_Images          ); fflush(0);         			
		printf("\nDataSet.Seq_start 		 		 = %d  ",DataSet.Seq_start 		           ); fflush(0);         			
		printf("\nDataSet.Seq_end 		 			 = %d  ",DataSet.Seq_end 		           ); fflush(0);         			
		printf("\nDataSet.File_Type 		 		 = %d  ",DataSet.File_Type 		           ); fflush(0);         			
		printf("\nDataSet.Data_Dim 		 		 	 = %d  ",DataSet.Data_Dim 		           ); fflush(0);         			
		printf("\nDataSet.Data_Type 		 		 = %d  ",DataSet.Data_Type 		           ); fflush(0);         			
		printf("\nDataSet.Width 			 		 = %d  ",DataSet.Width 			           ); fflush(0);         			
		printf("\nDataSet.Height 			 		 = %d  ",DataSet.Height 			       ); fflush(0);         			
		printf("\nDataSet.pxsize 	 			 = %f  ",DataSet.pxsize 	           ); fflush(0);         			
		printf("\nDataSet.pysize 	 			 = %f  ",DataSet.pysize 	           ); fflush(0);         			
		printf("\nDataSet.rxsize 		 		 = %f  ",DataSet.rxsize 		           ); fflush(0);         			
		printf("\nDataSet.rysize 		 		 = %f  ",DataSet.rysize 		           ); fflush(0);         			
		printf("\nDataSet.current_image	 		 	 = %d  ",DataSet.current_image	           ); fflush(0);         			
		printf("\nDataSet.min 			 			 = %d  ",DataSet.min 			           ); fflush(0);         			
		printf("\nDataSet.max 			 			 = %d  ",DataSet.max 			           ); fflush(0);         			
		printf("\nDataSet.min_f 			 		 = %f  ",DataSet.min_f 			           ); fflush(0);         			
		printf("\nDataSet.max_f 			 		 = %f  ",DataSet.max_f 			           ); fflush(0);         			
		printf("\nDataSet.TomographicType  		 	 = %d  ",DataSet.TomographicType           ); fflush(0);         			
		printf("\nDataSet.StrangeName 	 			 = %d  ",DataSet.StrangeName 	           ); fflush(0);         			
		printf("\nDataSet.Path			 			 = %s  ",DataSet.Path			           ); fflush(0);         			
		printf("\nDataSet.Name			 			 = %s  ",DataSet.Name			           ); fflush(0);         			
		printf("\nDataSet.Dark 			 		 	 = %d  ",DataSet.Dark 			           ); fflush(0);         			
		printf("\nDataSet.Izero 			 		 = %d  ",DataSet.Izero 			           ); fflush(0);         			
		printf("\nDataSet.CT_Step_todo 	 		 	 = %d  ",DataSet.CT_Step_todo 	           ); fflush(0);         			
		printf("\nDataSet.SCT_data_loaded	 		 = %d  ",DataSet.SCT_data_loaded	       ); fflush(0);         			
		printf("\nDataSet.arange 			 		 = %f  ",DataSet.arange 			       ); fflush(0);         			
		printf("\nDataSet.ringo_percent	 		 	 = %f  ",DataSet.ringo_percent	           ); fflush(0);         			
		printf("\nDataSet.ringo_percent2	 		 = %f  ",DataSet.ringo_percent2	           ); fflush(0);         			
		printf("\nDataSet.outlier_percent			 = %f  ",DataSet.outlier_percent		   ); fflush(0); 	   						
		printf("\nDataSet.Utility_Step_todo 		 = %d  ",DataSet.Utility_Step_todo 		   ); fflush(0); 						
		printf("\nDataSet.Seq_min					 = %d  ",DataSet.Seq_min				   ); fflush(0); 	   						
		printf("\nDataSet.Seq_max 					 = %d  ",DataSet.Seq_max 				   ); fflush(0); 		   			
		printf("\nDataSet.Seq_min_f 				 = %f  ",DataSet.Seq_min_f 				   ); fflush(0); 						
		printf("\nDataSet.Seq_max_f 				 = %f  ",DataSet.Seq_max_f 				   ); fflush(0); 						
		printf("\nDataSet.new_dataset 				 = %d  ",DataSet.new_dataset 			   ); fflush(0); 		   			
		printf("\nDataSet.metalartifact_percent 	 = %f  ",DataSet.metalartifact_percent 	   ); fflush(0); 						
		printf("\nDataSet.metalartifact_pixel 		 = %f  ",DataSet.metalartifact_pixel 	   ); fflush(0); 		   			
		printf("\nDataSet.metalartifact_down		 = %f  ",DataSet.metalartifact_down		   ); fflush(0); 						
		printf("\nDataSet.metalartifact_correction	 = %d  ",DataSet.metalartifact_correction  ); fflush(0);  					
		printf("\nDataSet.nslices					 = %d  ",DataSet.nslices				   ); fflush(0); 	   						
		printf("\nDataSet.nangles					 = %d  ",DataSet.nangles				   ); fflush(0); 	   						
		printf("\nDataSet.Image_U16					 = %d  ",DataSet.Image_U16				   ); fflush(0); 	   						
		printf("\nDataSet.Image_FL					 = %d  ",DataSet.Image_FL				   ); fflush(0); 	   						
		printf("\nDataSet.Image_U8					 = %d  ",DataSet.Image_U8				   ); fflush(0); 	   						
		printf("\n-----------------------------------------------------------------------------"); fflush(0);         			
																						

	return;
}


