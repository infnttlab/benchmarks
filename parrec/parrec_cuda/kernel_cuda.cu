#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "Global.h"



//-----------------------------------------------------------

__global__ void zeropaddingslice(float *dev_slice, int dominio){

	int contatore = blockIdx.x*blockDim.x + threadIdx.x;
	while (contatore < dominio){

		dev_slice[contatore] = 0.;
		contatore += blockDim.x*gridDim.x;
	}
}
//-----------------------------------------------------------

__global__ void creation_xStartEnd(int *d_x_start, int *d_x_end, float r, float l, float rxsize, int localCT_status, int localCT_x0, int localCT_x1, int width, int y_start, int y_end){

	int	xs, xe;
	float yf, radicale;

	int IdThreadInGrid = blockIdx.x*blockDim.x + threadIdx.x;
	int y = y_start + IdThreadInGrid; /*questo ci vuole perchè se no in ogni thread y partirebbe sempre da y_strat mentre ora
										th 0: y=y_start + 0
										th 1: y=y_start + 1
										th 2: y=y_start + 2
										th 3: y=y_start + 3
										     [...]        ...emulando così il for che non parte da zero */

//-----------------------------------------------------------------------------
// funzione per costringere x_start e x_end a stare dentro al cerchio r
// determinazione di x_start e x_end SENZA e CON localtomography
// questa funzione alloca i vettori x_start e x_end e li riempe di valori
//-----------------------------------------------------------------------------
	while(y <= y_end){

		yf = (float)y*rxsize -l;

		// cerchio r
		radicale = r*r-yf*yf;
		if( radicale > 0. ) {
			radicale = sqrt(radicale);
			xs = (int)( ( (l-radicale)/rxsize ) )+CUTSLICE;		// qui andrebbe + rxsize/2
			xe = (int)( ( (l+radicale)/rxsize ) )-CUTSLICE;		// qui andrebbe + rxsize/2
		} else {
			xs = (int)( l/rxsize )+CUTSLICE;
			xe = xs-CUTSLICE;
		}
		// controllo xs e xe nei limiti
		xs = (xs >= 0    ) ? xs : 0 ;
		xs = (xs < width ) ? xs : width-1 ;
		xe = (xe >= 0    ) ? xe : 0 ;
		xe = (xe < width ) ? xe : width-1 ;

		if(localCT_status == TRUE ) {
			xs = (localCT_x0 > xs) ? localCT_x0 : xs;		// localtomography
			xe = (localCT_x1 < xe) ? localCT_x1 : xe;			// localtomography
		}

		// si memorizzano x_start e x_end calcolati
		d_x_start[y] = xs;
		d_x_end[y] = xe;

		y += blockDim.x*gridDim.x;
	}
}
//-----------------------------------------------------------

__global__ void zp_d_signal(hipfftComplex *d_signal, int FFT_lenght){

	int IdThreadInGrid = blockIdx.x*blockDim.x + threadIdx.x;

	int contatore = IdThreadInGrid;
        while (contatore < FFT_lenght){

                d_signal[contatore].x = 0.;
                d_signal[contatore].y = 0.;
                contatore += blockDim.x*gridDim.x;
        }
}
//-----------------------------------------------------------
__global__ void initialize_signal(hipfftComplex *d_signal, float *d_Image_FL, int FFT_start, int width, int riga, int FFT_lenght, float *d_detector_weighing){

	int IdThreadInGrid = blockIdx.x*blockDim.x + threadIdx.x;

	int k = FFT_start + IdThreadInGrid;
	while(k<FFT_start+width){

		d_signal[k].x = d_Image_FL[riga+k-FFT_start];
		d_signal[k].x *= d_detector_weighing[k-FFT_start];
		k += blockDim.x*gridDim.x;
	}
}
//-----------------------------------------------------------

__global__ void convolution(hipfftComplex *d_signal, float *d_filter, int FFT_lenght){

	int contatore = blockIdx.x*blockDim.x + threadIdx.x;
	while (contatore < FFT_lenght){

		d_signal[contatore].x = d_signal[contatore].x * d_filter[contatore];
		d_signal[contatore].y= d_signal[contatore].y * d_filter[contatore];

		contatore += blockDim.x*gridDim.x;
	}
}
//-----------------------------------------------------------

__global__ void geometry_reconstruction(hipfftComplex *d_signal, float *dev_slice, float *d_Image_FL, int *d_x_start, int *d_x_end, float sod_quadro, float pxsize, float r, float rxsize, float xs, float ys, float teta_rad, float xd0, float xd1, float yd0, float yd1, float A, float B, float l, int caso, int nrays, int FFT_lenght, int FFT_start, int h, int localCT_status, int localCT_x0, int localCT_x1, int width, int y_start, int y_end, float sin_teta, float cos_teta, float xs_rot,float ys_rot,float xd0_rot,float xd1_rot,float yd0_rot,float yd1_rot,float xd_min,float xd_max,float yd_min,float yd_max, float xs_rot_quadro, float ys_rot_quadro, int riga){

	int iy = blockIdx.y*blockDim.y + threadIdx.y;
	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	int y = y_start + iy;
	int x = d_x_start[y] + ix;

	int ia, ib;
	float yf, xf, xT, yT, a, b, d, i_float, val, val_pesato_U, dA, dB;
	float U, U_quadro;


	if (y > y_end || x > d_x_end[y])
		return;

	// calcolo di yf
	yf = (float)y*rxsize -l;	// qui andrebbe + rxsize/2

	// calcolo di xf
	xf = (float)x*rxsize -l;		// qui andrebbe + rxsize/2

	// ------ si tracciano le rette ------ //

	// calcoliamo la retta per il punto xf,yf xs_rot,ys_rot
	// calcoliamo l'intersezione fra le due rette xT,yT, nell'ipotesi di detector non verticale
	// la retta del detector è y=Ax+B
	if( caso == 1 && xs_rot == xf ) {
		//ray_verticale=TRUE;   // se è TRUE vuol dire che b=0 e y=xs_rot
		xT = xs_rot;
		yT = xT*A+B;
	}
	else {
		//ray_verticale=FALSE;   // si calcolano normalmente a e b
		a = (ys_rot-yf)/(xs_rot-xf);
		b = yf - xf*a;
		if (caso == 1){
			xT = (b-B)/(A-a);
			yT = A*xT+B;
		}
		else {
			xT = xd0_rot;        // se è TRUE vuol dire che B=0 e y=xd_rot
			yT = xT*a+b;		 // se il detector è verticale non può esserlo anche il raggio in quanto per definizione
		}
	}

	// ora abbiamo calcolato xT,yT vediamo se si trova all'interno del detector
	if( xT >= xd_min && xT <= xd_max && yT >= yd_min && yT <= yd_max) {
		// c'è intersezione calcoliamo i_float
		d = (xT-xd0_rot)*(xT-xd0_rot) + (yT-yd0_rot)*(yT-yd0_rot);
		d = sqrt(d);
		i_float = d/pxsize;
		
		if( i_float >=0. && i_float < nrays) {  // qui ci va nrays x halfscan	
			// interpoliamo
			ia = (int) i_float;
			ib = ia+1;
			dB = i_float - (float)ia;
			dA = 1. - dB;
	
			// calcoliamo il valore interpolato da associare a questo punto
			// qui eventualmente saltare se //						val =0.;
			if( d_Image_FL[riga+ia] > 0. && ib < nrays && d_Image_FL[riga+ib] > 0. ) {
				val = d_signal[ia+FFT_start].x * dA;
				val += d_signal[ib+FFT_start].x * dB;
	
				// peso per la distanza dalla proiezione sulla retta della sorgente ---W3--- EQUISPAZIATI
				U = (sod_quadro+xs_rot_quadro+ys_rot_quadro-2.*xs_rot*xf-2.*ys_rot*yf)/sod_quadro;
				U_quadro = U * U;
				val_pesato_U = val/U_quadro;
				// fattore 1/N della FFT
				val_pesato_U /= (float) FFT_lenght;
	
				// associazione del punto nella slice_rank
				dev_slice[y*width+x] += val_pesato_U;
			}
		} // fine di se i_float valori trovati sono validi
	} // fine di controllo su xT e yT
}
//-----------------------------------------------------------



