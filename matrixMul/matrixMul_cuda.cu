#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include "helper_string.h"

void printMatrix(int row, int col, float *matrix){
        int i, j;
        for(i=0; i<row; i++){
                for(j=0; j<col; j++){
                      printf("%f ", matrix[i*col+j]);
                }
                printf("\n");
        }
}

int help_func(){
        printf("\nUsage:   -rA=RowsA(d:512)     -cA=ColumnsA(d:512)  -cB=ColumnsB(d:512) | matrix(row,col), ColumnsA = RowsB\n");
	printf("         -db=DimBlock(d:16)                             | DimBlock(in threads): [1-32], block(DimBlock, DimBlock)\n");
        printf("         -w=WarmUpData(d:0)\n");
        printf("         -v=Verbose(d:0)\n\n");
       // printf("Default: A = (512,512) B = (512,512); DIM_BLOCK = 16; WARMUP = 0; VERBOSE = 0\n\n");

        return 0;
}


__global__
void matrixFillKernel(int rowMax, int colMax,  float *d_matrix){
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int i = row*colMax+col;
        if(row<rowMax && col<colMax){
                d_matrix[i] = 0.1f;
        }
}

__global__
void matrixMulKernel(int row_a, int col_a, int col_b, float* d_matrix_a, float* d_matrix_b, float* d_matrix_c){
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        int k;

	d_matrix_c[row*col_b + col] = 0.f;
        if(col<col_b && row<row_a){
                for(k=0; k<col_a; k++){
                        d_matrix_c[row*col_b + col] += d_matrix_a[row*col_a + k] * d_matrix_b[k*col_b + col];
                }
        }
}


int main(int argc, char **argv){
        int val_returned = 0;
        if(argc == 2 && ( (strcmp(argv[1], "--help")==0) || (strcmp(argv[1], "-h")==0) )){
                val_returned = help_func();
        }
        else{
                int row_a = 512, col_a = 512, col_b = 512;
		int debug = 0, perf = 0;
		int dimBlock = 16;

		if (checkCmdLineFlag(argc, (const char **)argv, "rA")){
                        row_a = getCmdLineArgumentInt(argc, (const char **)argv, "rA");
                }
                if (checkCmdLineFlag(argc, (const char **)argv, "cA")){
                        col_a = getCmdLineArgumentInt(argc, (const char **)argv, "cA");
                }
                if (checkCmdLineFlag(argc, (const char **)argv, "cB")){
                        col_b = getCmdLineArgumentInt(argc, (const char **)argv, "cB");
                }
                if (checkCmdLineFlag(argc, (const char **)argv, "w")){
                        perf = getCmdLineArgumentInt(argc, (const char **)argv, "w");
                }
                if (checkCmdLineFlag(argc, (const char **)argv, "v")){
                        debug = getCmdLineArgumentInt(argc, (const char **)argv, "v");
                }
		if (checkCmdLineFlag(argc, (const char **)argv, "db")){
                        dimBlock = getCmdLineArgumentInt(argc, (const char **)argv, "db");
                }


                if(dimBlock>32){
                        val_returned =  help_func();
                }
                else{
                        hipEvent_t start_all, end_all,
					start_fill, end_fill,
					start_mm, end_mm,
					start_free, end_free;

			hipEventCreate(&start_all);
                        hipEventCreate(&end_all);
			hipEventCreate(&start_fill);
                        hipEventCreate(&end_fill);
			hipEventCreate(&start_mm);
                        hipEventCreate(&end_mm);
			hipEventCreate(&start_free);
                        hipEventCreate(&end_free);

                        hipEventRecord(start_all, 0);

                        float *matrix_a = (float*)malloc(row_a*col_a * sizeof(float));
                        float *matrix_b = (float*)malloc(col_a*col_b * sizeof(float));
                        float *matrix_c = (float*)malloc(row_a*col_b * sizeof(float));

                        float *d_matrix_a;
                        float *d_matrix_b;
                        float *d_matrix_c;

                        hipMalloc((void**)&d_matrix_a, (row_a*col_a * sizeof(float)));
                        hipMalloc((void**)&d_matrix_b, (col_a*col_b * sizeof(float)));
                        hipMalloc((void**)&d_matrix_c, (row_a*col_b * sizeof(float)));

                        dim3 block(dimBlock,dimBlock);

                        //matrix with theadsPerRow = row_a threadsPerCol = col_a
                        //              dimBlock.x = col                        dimBlock.y = row
                        dim3 gridA( (int)ceil(col_a/(float)dimBlock) , (int)ceil(row_a/(float)dimBlock) );
                        dim3 gridB( (int)ceil(col_b/(float)dimBlock) , (int)ceil(col_a/(float)dimBlock)  );
                        dim3 gridC( (int)ceil(col_b/(float)dimBlock) , (int)ceil(row_a/(float)dimBlock)  );

			hipEventRecord(end_all, 0);
                        hipEventSynchronize(end_all);
			float timeAlloc;
                        hipEventElapsedTime(&timeAlloc, start_all, end_all);

                        printf("\n### Matrix A = (%d,%d); Matrix B = (%d,%d); AxB = (%d,%d);\n",
					row_a, col_a, col_a, col_b, row_a, col_b);
                        printf("### dimBlock = %d; gridA(%d,%d); gridB(%d,%d); gridC(%d,%d);\n",
					dimBlock,
					(int)ceil(col_a/(float)dimBlock),(int)ceil(row_a/(float)dimBlock),
					(int)ceil(col_b/(float)dimBlock),(int)ceil(col_a/(float)dimBlock),
					(int)ceil(col_b/(float)dimBlock),(int)ceil(row_a/(float)dimBlock)
				);

			if(debug){                        
				int col_gA = (int)ceil(col_a/(float)dimBlock);
        	                int row_gA = (int)ceil(row_a/(float)dimBlock);
                	        int col_gB = (int)ceil(col_b/(float)dimBlock);
                        	int row_gB = (int)ceil(col_a/(float)dimBlock);
	                        int col_gC = (int)ceil(col_b/(float)dimBlock);
        	                int row_gC = (int)ceil(row_a/(float)dimBlock);

	                        int totThA = col_gA*row_gA*dimBlock*dimBlock;
        	                int totThB = col_gB*row_gB*dimBlock*dimBlock;
                	        int totThC = col_gC*row_gC*dimBlock*dimBlock;

	                        printf("\n******************** TEST ***********************\n");
        	                printf("- totThA = %d VS totElA = %d\n", totThA,col_a*row_a);
                	        printf("- totThB = %d VS totElB = %d\n", totThB,col_a*col_b);
	                        printf("- totThC = %d VS totElC = %d\n", totThC,col_b*row_a);
        	                printf("*************************************************\n");
			}


			hipEventRecord(start_fill, 0);

                        matrixFillKernel<<<gridA,block>>>(row_a,col_a,d_matrix_a);
                        matrixFillKernel<<<gridB,block>>>(col_a,col_b,d_matrix_b);

			hipEventRecord(end_fill, 0);
                        hipEventSynchronize(end_fill);
			float timeFill;
                        hipEventElapsedTime(&timeFill, start_fill, end_fill);

			if(perf){
				//Performs warmup operation
				printf("\nPreforming warmup...\n");
				matrixMulKernel<<<gridC,block>>>(row_a,col_a,col_b,d_matrix_a,d_matrix_b,d_matrix_c);
			}

			printf("\nComputing matrix multimplication...\n");
			hipEventRecord(start_mm, 0);
                        matrixMulKernel<<<gridC,block>>>(row_a,col_a,col_b,d_matrix_a,d_matrix_b,d_matrix_c);

			hipEventRecord(end_mm, 0);
                        hipEventSynchronize(end_mm);
			float timeMtxMul;
                        hipEventElapsedTime(&timeMtxMul, start_mm, end_mm);

			if(debug == 2){
                        	hipMemcpy(matrix_a, d_matrix_a, (row_a*col_a)*sizeof(float), hipMemcpyDeviceToHost);
                        	hipMemcpy(matrix_b, d_matrix_b, (col_a*col_b)*sizeof(float), hipMemcpyDeviceToHost);
                        	hipMemcpy(matrix_c, d_matrix_c, (row_a*col_b)*sizeof(float), hipMemcpyDeviceToHost);
			}


			hipEventRecord(start_free, 0);

                        hipFree(d_matrix_c); hipFree(d_matrix_a); hipFree(d_matrix_b);

			if(debug == 2){
		                //print all matrix:
        		        printf("\n## Matrix A:\n");
                		printMatrix(row_a, col_a, matrix_a);
	                	printf("\n## Matrix B:\n");
	        	        printMatrix(col_a, col_b, matrix_b);
        	        	printf("\n## Matrix C:\n");
	        	        printMatrix(row_a, col_b, matrix_c);
        	        }

			free(matrix_a); free(matrix_b); free(matrix_c);			

			hipEventRecord(end_free, 0);
                        hipEventSynchronize(end_free);
			float timeFree;
                        hipEventElapsedTime(&timeFree, start_free, end_free);

                        printf("\nTerminated.\n");

			double flops4mtxmul = 2.0*(double)row_a*(double)col_a*(double)col_b;
			double gigaFlops = (flops4mtxmul * 1.0e-9f) / (timeMtxMul / 1000.0f);

		//	printf("\nPerformance: %f GFlop/s; Time: %f ms; Flop: %f\n\n", gigaFlops, timeMtxMul, flops4mtxmul);
			if(debug){
                        	printf("\nDimBlock: %d,  Flop: %.0f,  GFlop: %f GFlop/s,  Time_mtxMul: %f s\n",
                                	dimBlock, flops4mtxmul, gigaFlops, timeMtxMul/1000.0f);
				printf("Time_tot:");
				printf("\nA. timeAllocation: %f s;\nB. timeComputation: %f s (fill: %f s, matrixMul: %f s);\nC. timeFree: %f s;\nD. TOTAL: %f s\n\n",
					timeAlloc/1000.0f,
					(timeFill+timeMtxMul)/1000.0f,
					timeFill/1000.0f,
					timeMtxMul/1000.0f,
					timeFree/1000.0f,
					(timeAlloc+timeFill+timeMtxMul+timeFree)/1000.0f
				);
			}
                	else
                        	 printf("\n%d %.0f %f %f %f\n\n",
                                        dimBlock, flops4mtxmul, gigaFlops, timeMtxMul/1000.0f, (timeAlloc+timeFill+timeMtxMul+timeFree)/1000.0f);
                }
        }
        return val_returned;
}
