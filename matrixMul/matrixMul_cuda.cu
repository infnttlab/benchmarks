#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>

void printMatrix(int row, int col, float *matrix){
        int i, j;
        for(i=0; i<row; i++){
                for(j=0; j<col; j++){
                      printf("%f ", matrix[i*col+j]);
                }
                printf("\n");
        }
}

int help_func(){
        printf("\nUsage: ./a.out <ROW_A> <COL_A> <COL_B> <DIM_BLOCK>\n");
        printf("Where:   MATRIX(ROW, COL)  and  <COL_A> == <ROW_B>\n");
        printf("         DIM_BLOCK: [1-32]; BLOCK(dimBlock, dimBlock)\n\n");
        printf("Default: A = (512,512); B = (512,512); DIM_BLOCK = 16;\n\n");

        return 0;
}

__global__
void matrixFillKernel(int rowMax, int colMax,  float *d_matrix){
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int i = row*colMax+col;
        if(row<rowMax && col<colMax){
                d_matrix[i] = 0.1f;
        }
}

__global__
void matrixMulKernel(int row_a, int col_a, int col_b, float* d_matrix_a, float* d_matrix_b, float* d_matrix_c){
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        int k;

	d_matrix_c[row*col_b + col] = 0.f;
        if(col<col_b && row<row_a){
                for(k=0; k<col_a; k++){
                        d_matrix_c[row*col_b + col] += d_matrix_a[row*col_a + k] * d_matrix_b[k*col_b + col];
                }
        }
}


int main(int argc, char **argv){
        int val_returned = 0;
        if(argc == 2 && ( (strcmp(argv[1], "--help")==0) || (strcmp(argv[1], "-h")==0) )){
                val_returned = help_func();
        }
        else{
                int row_a = 512, col_a = 512, col_b = 512;
		int debug = 0;
                if(argc >= 2){
                        // change ROW_A
                        row_a = atoi(argv[1]);

                        if(argc >= 3){
                                // change ROW_A COL_A and ROW_B, where COL_A = ROW_B
                                col_a = atoi(argv[2]);
                                if(argc >= 4){
                                        col_b = atoi(argv[3]);
					if(argc == 5)
						debug = atoi(argv[4]);
                                }
                        }
                }

                int dimBlock = 16;
                if(argc >= 5){
                        dimBlock = atoi(argv[4]);
                }

                if(dimBlock>32){
                        val_returned =  help_func();
                }
                else{
                        struct timeval tstart, tstop;
                        double elapsed = 0.f;

                        hipEvent_t startCUDA, stopCUDA;
                        double timeCUDA;

                        hipEventCreate(&startCUDA);
                        hipEventCreate(&stopCUDA);

                        gettimeofday(&tstart,NULL);
                        hipEventRecord(startCUDA, 0);

                        float *matrix_a = (float*)malloc(row_a*col_a * sizeof(float));
                        float *matrix_b = (float*)malloc(col_a*col_b * sizeof(float));
                        float *matrix_c = (float*)malloc(row_a*col_b * sizeof(float));

                        float *d_matrix_a;
                        float *d_matrix_b;
                        float *d_matrix_c;

                        hipMalloc((void**)&d_matrix_a, (row_a*col_a * sizeof(float)));
                        hipMalloc((void**)&d_matrix_b, (col_a*col_b * sizeof(float)));
                        hipMalloc((void**)&d_matrix_c, (row_a*col_b * sizeof(float)));

                        dim3 block(dimBlock,dimBlock);

                        //matrix with theadsPerRow = row_a threadsPerCol = col_a
                        //              dimBlock.x = col                        dimBlock.y = row
                        dim3 gridA( (int)ceil(col_a/(float)dimBlock) , (int)ceil(row_a/(float)dimBlock) );
                        dim3 gridB( (int)ceil(col_b/(float)dimBlock) , (int)ceil(col_a/(float)dimBlock)  );
                        dim3 gridC( (int)ceil(col_b/(float)dimBlock) , (int)ceil(row_a/(float)dimBlock)  );

			if(debug){
                        	printf("\n### Matrix A = (%d,%d); Matrix B = (%d,%d); AxB = (%d,%d);\n",
					row_a, col_a, col_a, col_b, row_a, col_b);
                        	printf("### dimBlock = %d; gridA(%d,%d); gridB(%d,%d); gridC(%d,%d);\n",
					dimBlock, (int)ceil(col_a/(float)dimBlock),(int)ceil(row_a/(float)dimBlock),
					(int)ceil(col_b/(float)dimBlock),(int)ceil(col_a/(float)dimBlock),
					(int)ceil(col_b/(float)dimBlock) , (int)ceil(row_a/(float)dimBlock));
                        
				int col_gA = (int)ceil(col_a/(float)dimBlock);
        	                int row_gA = (int)ceil(row_a/(float)dimBlock);
                	        int col_gB = (int)ceil(col_b/(float)dimBlock);
                        	int row_gB = (int)ceil(col_a/(float)dimBlock);
	                        int col_gC = (int)ceil(col_b/(float)dimBlock);
        	                int row_gC = (int)ceil(row_a/(float)dimBlock);

	                        int totThA = col_gA*row_gA*dimBlock*dimBlock;
        	                int totThB = col_gB*row_gB*dimBlock*dimBlock;
                	        int totThC = col_gC*row_gC*dimBlock*dimBlock;

	                        printf("\n******************** TEST ***********************\n");
        	                printf("- totThA = %d VS totElA = %d\n", totThA,col_a*row_a);
                	        printf("- totThB = %d VS totElB = %d\n", totThB,col_a*col_b);
	                        printf("- totThC = %d VS totElC = %d\n", totThC,col_b*row_a);
        	                printf("*************************************************\n");
			}

                        matrixFillKernel<<<gridA,block>>>(row_a,col_a,d_matrix_a);
                        matrixFillKernel<<<gridB,block>>>(col_a,col_b,d_matrix_b);

                        matrixMulKernel<<<gridC,block>>>(row_a,col_a,col_b,d_matrix_a,d_matrix_b,d_matrix_c);

			if(debug){
                        	hipMemcpy(matrix_a, d_matrix_a, (row_a*col_a)*sizeof(float), hipMemcpyDeviceToHost);
                        	hipMemcpy(matrix_b, d_matrix_b, (col_a*col_b)*sizeof(float), hipMemcpyDeviceToHost);
                        	hipMemcpy(matrix_c, d_matrix_c, (row_a*col_b)*sizeof(float), hipMemcpyDeviceToHost);
			}

                        hipFree(d_matrix_c); hipFree(d_matrix_a); hipFree(d_matrix_b);

			if(debug){
		                //print all matrix:
        		        printf("\n## Matrix A:\n");
                		printMatrix(row_a, col_a, matrix_a);
	                	printf("\n## Matrix B:\n");
	        	        printMatrix(col_a, col_b, matrix_b);
        	        	printf("\n## Matrix C:\n");
	        	        printMatrix(row_a, col_b, matrix_c);
        	        }

			free(matrix_a); free(matrix_b); free(matrix_c);			

                        hipEventRecord(stopCUDA, 0);
                        hipEventSynchronize(stopCUDA);
                        hipEventElapsedTime(&timeCUDA, startCUDA, stopCUDA);

                        gettimeofday(&tstop,NULL);
                        printf("\nTerminated.\n");
                        elapsed = (tstop.tv_sec - tstart.tv_sec) + ((tstop.tv_usec - tstart.tv_usec)/1000000.0);

                        printf("Data processing in %f s using \"gettimeofday\" and %f ms using \"CUDA Events\".\n\n",
				elapsed, timeCUDA);
                }
        }
        return val_returned;
}
